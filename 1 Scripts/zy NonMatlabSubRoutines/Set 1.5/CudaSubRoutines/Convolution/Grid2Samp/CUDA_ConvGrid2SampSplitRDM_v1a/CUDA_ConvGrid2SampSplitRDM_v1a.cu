
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///  	- switch to lrint
///		(still to-do: remove (j<DatLen) check)
///==========================================================

extern "C" void ConvGrid2SampSplitRDM(int Count, size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);

													
///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(double* dSDat, double* dGDat, double* dKx, double* dKy, double* dKz, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
int xflr,yflr,zflr;
double DatVal,KernVal,CrtVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = 0;
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
    zflr = __double2int_rd(dKz[j]);   
	//xF = lround(iKern*(dKx[j]-xflr));     		// halfway rounded away from zero      
    //yF = lround(iKern*(dKy[j]-yflr)); 
    //zF = lround(iKern*(dKz[j]-zflr));	
	xF = lrint(iKern*(dKx[j]-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(dKy[j]-yflr)); 
    zF = lrint(iKern*(dKz[j]-zflr));
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}
	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz + lrint(fabsf(zF-(c*iKern)))*KernSz*KernSz];
				CrtVal = dGDat[(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)];
				DatVal += KernVal*CrtVal;							
				}
			}
		} 
	dSDat[j] = DatVal;
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitRDM(int Count, size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
    int PtsPerDevice = int(ceil(double(DatLen)/double(Count)));
	int bpg = int(ceil(double(PtsPerDevice)/double(tpb)));  
    for (int n=0; n<Count; n++){	
        hipSetDevice(n);
        dSDat = (double*)(HSampDat[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dGDat = (double*)HGrdDat[n];
        dKx = (double*)(HKx[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKy = (double*)(HKy[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));	
        dKz = (double*)(HKz[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKern = (double*)HKern[n];	                
        Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,PtsPerDevice,KernSz,iKern,chW);
    }
	hipDeviceSynchronize();												// make sure finished	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}	
	

