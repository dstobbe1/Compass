
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- RS_v1c start
///==========================================================

extern "C" void ConvGrid2SampSplitRD2D(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);

													
///=====================================================
/// Conv2D (kernel)					
///=====================================================
__global__ void Conv2D(double* dSDat, double* dGDat, double* dKx, double* dKy, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF;
//double xF,yF;
int xflr,yflr;
double DatVal,KernVal,CrtVal;
int j,a,b;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = 0;
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]); 
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
	for(b=-chW; b<=chW+1; b++) {
		for(a=-chW; a<=chW+1; a++) {
			KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz];
			CrtVal = dGDat[(xflr+a-1)+((yflr+b-1)*GrdDatSz)];
			DatVal += KernVal*CrtVal;							
			}
		}
	dSDat[j] = DatVal;
	}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitRD2D(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDat,*dGDat,*dKx,*dKy,*dKern;
	dSDat = (double*)(*HSampDat+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv2D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished		
}
							

