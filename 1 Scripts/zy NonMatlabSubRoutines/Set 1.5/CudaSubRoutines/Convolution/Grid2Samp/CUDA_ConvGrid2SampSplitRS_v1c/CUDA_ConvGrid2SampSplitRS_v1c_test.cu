
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- function accuracy fixes
///==========================================================

extern "C" void ConvGrid2SampSplitRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDat, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
//float xrmd,yrmd,zrmd;
//float fxflr,fyflr,fzflr;
int xflr,yflr,zflr;
int xF,yF,zF;
float DatVal,KernVal,CrtVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = 0;
	KernVal = 0;
	CrtVal = 0;
    //--------------------------------
	//fxflr = floorf(dKx[j]);			// original - not accurate enough
    //fyflr = floorf(dKy[j]);
    //fzflr = floorf(dKz[j]);
	xflr = __float2int_rd(dKx[j]);
    yflr = __float2int_rd(dKy[j]);
    zflr = __float2int_rd(dKz[j]);   
	//--------------------------------
	
	//--------------------------------
	//xrmd = dKx[j]-fxflr;
    //yrmd = dKy[j]-fyflr;
    //zrmd = dKz[j]-fzflr;
	//xF = lrintf(iKern*xrmd);      		// original  (change = no impact)               
    //yF = lrintf(iKern*yrmd); 
    //zF = lrintf(iKern*zrmd);
	//xF = lrintf(iKern*(dKx[j]-xflr));                  
    //yF = lrintf(iKern*(dKy[j]-yflr)); 
    //zF = lrintf(iKern*(dKz[j]-zflr));
	xF = lroundf(iKern*(dKx[j]-xflr));       // - round up (still has error - probably precision?)             
    yF = lroundf(iKern*(dKy[j]-yflr)); 
    zF = lroundf(iKern*(dKz[j]-zflr));	
	//---------------------------------

	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
	}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
	}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
	}

	//-------------------------------
	//xflr = __float2int_rd(fxflr);			// original		
	//yflr = __float2int_rd(fyflr);
	//zflr = __float2int_rd(fzflr);
	//xflr = lrintf(fxflr);					// this change has no effect	
	//yflr = lrintf(fyflr);
	//zflr = lrintf(fzflr);	
	//xflr = __float2int_ru(fxflr);			// this change has no effect  (becuase already rounded)	
	//yflr = __float2int_ru(fyflr);
	//zflr = __float2int_ru(fzflr);
	//-------------------------------
 

	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrintf(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];
				//KernVal = dKern[__float2int_rd(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];   % no effect
				//KernVal = dKern[lroundf(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];
				CrtVal = dGDat[(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)];
				DatVal += KernVal*CrtVal;
				//DatVal += CrtVal;
				//DatVal += KernVal;
			}
		}
	} 
	dSDat[j] = DatVal;
	//dSDat[j] = xF;
	//dSDat[j] = xflr;	
	//dSDat[j] = dKern[lrintf(fabsf(xF-((2)*iKern))+(fabsf(yF-((2)*iKern))*KernSz)+(fabsf(zF-((2)*iKern))*KernSz*KernSz))];
	//dSDat[j] = dKern[17056480];
}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	float *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDat = (float*)(*HSampDat+SampDatAdr*sizeof(float));
	dGDat = (float*)*HGrdDat;
	dKx = (float*)(*HKx+SampDatAdr*sizeof(float));
	dKy = (float*)(*HKy+SampDatAdr*sizeof(float));	
	dKz = (float*)(*HKz+SampDatAdr*sizeof(float));
	dKern = (float*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(DatLen)/float(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished		
}
							

