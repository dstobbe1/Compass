
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- large kern referencing fix (float related)
///==========================================================

extern "C" void ConvGrid2SampSplitRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDat, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
//float xF,yF,zF;
int xflr,yflr,zflr;
float DatVal,KernVal,CrtVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = 0;
	xflr = __float2int_rd(dKx[j]);
    yflr = __float2int_rd(dKy[j]);
    zflr = __float2int_rd(dKz[j]);   
	xF = lroundf(iKern*(dKx[j]-xflr));     	// lroundf necessary to match 'mex' and 'matlab'         
    yF = lroundf(iKern*(dKy[j]-yflr)); 
    zF = lroundf(iKern*(dKz[j]-zflr));	
	//xF = roundf(iKern*(dKx[j]-xflr));      // same result if xF is 'int' or 'float'
    //yF = roundf(iKern*(dKy[j]-yflr)); 
    //zF = roundf(iKern*(dKz[j]-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}
	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz + lrintf(fabsf(zF-(c*iKern)))*KernSz*KernSz];		// individual conversion necessary (i.e. for 'z') because number will be too big for float to handle 
				CrtVal = dGDat[(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)];
				DatVal += KernVal*CrtVal;							// looks like some small precision error in this command when compared to 'mex' and 'matlab'
				}
			}
		} 
	dSDat[j] = DatVal;
	}

}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitRS(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	float *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDat = (float*)(*HSampDat+SampDatAdr*sizeof(float));
	dGDat = (float*)*HGrdDat;
	dKx = (float*)(*HKx+SampDatAdr*sizeof(float));
	dKy = (float*)(*HKy+SampDatAdr*sizeof(float));	
	dKz = (float*)(*HKz+SampDatAdr*sizeof(float));
	dKern = (float*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(DatLen)/float(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished		
}
							

