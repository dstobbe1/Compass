
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		- SampDat now interleaved complex
///		(still to-do:)
///         - remove (j<DatLen) check)
///         - what about xflr and xF as inputs (calculate only once)
///==========================================================

extern "C" void ConvGrid2SampSplitCSM(int Count, size_t *HSampDatC, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, int SampDatAdrC, char* Error);

													
///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDatC, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
int xflr,yflr,zflr;
float DatValR,DatValI,KernVal,CrtValR,CrtValI;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = 0;
    DatValI = 0;
	xflr = __float2int_rd(dKx[j]);
    yflr = __float2int_rd(dKy[j]);
    zflr = __float2int_rd(dKz[j]);   
	xF = lrint(iKern*(dKx[j]-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(dKy[j]-yflr)); 
    zF = lrint(iKern*(dKz[j]-zflr));
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}
	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz + lrintf(fabsf(zF-(c*iKern)))*KernSz*KernSz];
				CrtValR = dGDat[((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				CrtValI = dGDat[((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				DatValR += KernVal*CrtValR;
				DatValI += KernVal*CrtValI;							
				}
			}
		} 
	dSDatC[2*j] = DatValR;
	dSDatC[2*j+1] = DatValI;
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitCSM(int Count, size_t *HSampDatC, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, int SampDatAdrC, char* Error){

	float *dSDatC,*dGDat,*dKx,*dKy,*dKz,*dKern;
	int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
    int PtsPerDevice = int(ceil(float(DatLen)/float(Count)));
	int bpg = int(ceil(float(PtsPerDevice)/float(tpb)));  
    for (int n=0; n<Count; n++){	
        hipSetDevice(n);
        dSDatC = (float*)(HSampDatC[n]+(SampDatAdrC+n*PtsPerDevice*2)*sizeof(float));
        dGDat = (float*)HGrdDat[n];
        dKx = (float*)(HKx[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));
        dKy = (float*)(HKy[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));	
        dKz = (float*)(HKz[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));
        dKern = (float*)HKern[n];	                
        Conv3D<<<bpg,tpb>>>(dSDatC,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,PtsPerDevice,KernSz,iKern,chW);
    }
	hipDeviceSynchronize();												// make sure finished	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}	
	

