
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		- SampDat now interleaved complex
///		(still to-do:)
///         - remove (j<DatLen) check)
///         - what about xflr and xF as inputs (calculate only once)
///==========================================================

extern "C" void ConvGrid2SampSplitCDM(int Count, size_t *HSampDatC, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, int SampDatAdrC, char* Error);

													
///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(double* dSDatC, double* dGDat, double* dKx, double* dKy, double* dKz, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
int xflr,yflr,zflr;
double DatValR,DatValI,KernVal,CrtValR,CrtValI;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = 0;
    DatValI = 0;
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
    zflr = __double2int_rd(dKz[j]);   
	xF = lrint(iKern*(dKx[j]-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(dKy[j]-yflr)); 
    zF = lrint(iKern*(dKz[j]-zflr));
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}
	for(c=-chW; c<=chW+1; c++) {
		for(b=-chW; b<=chW+1; b++) {
			for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernSz + lrintf(fabsf(zF-(c*iKern)))*KernSz*KernSz];
				CrtValR = dGDat[((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				CrtValI = dGDat[((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)];
				DatValR += KernVal*CrtValR;
				DatValI += KernVal*CrtValI;							
				}
			}
		} 
	dSDatC[2*j] = DatValR;
	dSDatC[2*j+1] = DatValI;
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvGrid2SampSplitCDM(int Count, size_t *HSampDatC, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, int SampDatAdrC, char* Error){

	double *dSDatC,*dGDat,*dKx,*dKy,*dKz,*dKern;
	int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
    int PtsPerDevice = int(ceil(double(DatLen)/double(Count)));
	int bpg = int(ceil(double(PtsPerDevice)/double(tpb)));  
    for (int n=0; n<Count; n++){	
        hipSetDevice(n);
        dSDatC = (double*)(HSampDatC[n]+(SampDatAdrC+n*PtsPerDevice*2)*sizeof(double));
        dGDat = (double*)HGrdDat[n];
        dKx = (double*)(HKx[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKy = (double*)(HKy[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));	
        dKz = (double*)(HKz[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(double));
        dKern = (double*)HKern[n];	                
        Conv3D<<<bpg,tpb>>>(dSDatC,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,PtsPerDevice,KernSz,iKern,chW);
    }
	hipDeviceSynchronize();												// make sure finished	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}	
	

