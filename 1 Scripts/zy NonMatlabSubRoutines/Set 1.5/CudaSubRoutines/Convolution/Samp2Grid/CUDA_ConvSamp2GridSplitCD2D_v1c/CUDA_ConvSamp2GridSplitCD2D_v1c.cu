
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- RS_v1c start
///==========================================================

extern "C" void ConvSamp2GridSplitCD2D(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


__device__ double atomicAddD(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
		} while (assumed != old);
	return __longlong_as_double(old);
}							

///=====================================================
/// Conv2D (kernel)					
///=====================================================
__global__ void Conv2D(double* dSDatR, double* dSDatI, double* dGDat, double* dKx, double* dKy, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF;
//double xF,yF;
int xflr,yflr;
double DatValR,DatValI,KernVal;
int j,a,b;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = dSDatR[j];
    DatValI = dSDatI[j];	
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}	
	for(b=-chW; b<=chW+1; b++) {
		for(a=-chW; a<=chW+1; a++) {
			KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz];		
			atomicAddD((dGDat+((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)),(KernVal*DatValR));
			atomicAddD((dGDat+((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)),(KernVal*DatValI));
			}
		}
	}  
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitCD2D(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKern;
	dSDatR = (double*)(*HSampDatR+SampDatAdr*sizeof(double));
	dSDatI = (double*)(*HSampDatI+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv2D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished	
}
							
