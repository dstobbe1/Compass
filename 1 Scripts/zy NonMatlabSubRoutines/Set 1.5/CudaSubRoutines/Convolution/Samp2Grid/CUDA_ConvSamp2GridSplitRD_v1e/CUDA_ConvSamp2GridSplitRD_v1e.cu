
#include <hip/hip_runtime.h>
///==========================================================
/// (v1e)
///		- Recompile with CUDA 9.0
///==========================================================

extern "C" void ConvSamp2GridSplitRD(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


__device__ double atomicAddD(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
		} while (assumed != old);
	return __longlong_as_double(old);
}							

///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(double* dSDat, double* dGDat, double* dKx, double* dKy, double* dKz, double* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
//double xF,yF,zF;
int xflr,yflr,zflr;
double DatVal,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = dSDat[j];
	xflr = __double2int_rd(dKx[j]);
    yflr = __double2int_rd(dKy[j]);
    zflr = __double2int_rd(dKz[j]);   
	xF = lround(iKern*(dKx[j]-xflr));     	// lround necessary to match 'mex' and 'matlab'         
    yF = lround(iKern*(dKy[j]-yflr)); 
    zF = lround(iKern*(dKz[j]-zflr));	
	//xF = round(iKern*(dKx[j]-xflr));      
    //yF = round(iKern*(dKy[j]-yflr)); 
    //zF = round(iKern*(dKz[j]-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}		
    for(c=-chW; c<=chW+1; c++) {
        for(b=-chW; b<=chW+1; b++) {
            for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz + lrint(fabsf(zF-(c*iKern)))*KernSz*KernSz];		
				atomicAddD((dGDat+(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)),(KernVal*DatVal));
				}
			}
		}  
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitRD(size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	double *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDat = (double*)(*HSampDat+SampDatAdr*sizeof(double));
	dGDat = (double*)*HGrdDat;
	dKx = (double*)(*HKx+SampDatAdr*sizeof(double));
	dKy = (double*)(*HKy+SampDatAdr*sizeof(double));	
	dKz = (double*)(*HKz+SampDatAdr*sizeof(double));
	dKern = (double*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(DatLen)/double(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

