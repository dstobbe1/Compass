
#include <hip/hip_runtime.h>
///==========================================================
/// (v2a)
///		- 
///==========================================================

extern "C" void ConvSamp2GridComplex(size_t *GpuNum, size_t *HSampDat, size_t *HReconInfo, size_t *HKernel, size_t *HImageMatrix,
                                        size_t *SampDatMemDims, size_t *KernelMemDims, size_t *ImageMatrixMemDims, 
                                        size_t *iKern, size_t *KernHw, char *Error);
						

///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSampDat, float* dReconInfo, float* dKernel, float* dImageMatrix, 
					   int KernelMemDims, int ImageMatrixMemDims0,  int ImageMatrixMemDims1, int iKern, int KernHw, int SampDatLength)
{
float Kx,Ky,Kz,Sdc;
int xF,yF,zF;
int xflr,yflr,zflr;
float DatValR,DatValI;
float KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < SampDatLength) {

    Kx = dReconInfo[j];
    Ky = dReconInfo[SampDatLength+j];
    Kz = dReconInfo[2*SampDatLength+j];
    Sdc = dReconInfo[3*SampDatLength+j];
    
    DatValR = dSampDat[2*j]*Sdc;
    DatValI = dSampDat[2*j+1]*Sdc;	
    
	xflr = __float2int_rd(Kx);
    yflr = __float2int_rd(Ky);
    zflr = __float2int_rd(Kz);   
	xF = lrintf(iKern*(Kx-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrintf(iKern*(Ky-yflr)); 
    zF = lrintf(iKern*(Kz-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}	
       
    for(c=-KernHw; c<=KernHw+1; c++) {
        for(b=-KernHw; b<=KernHw+1; b++) {
            for(a=-KernHw; a<=KernHw+1; a++) {
				KernVal = dKernel[lrintf(fabsf(xF-(a*iKern))) + lrintf(fabsf(yF-(b*iKern)))*KernelMemDims + lrintf(fabsf(zF-(c*iKern)))*KernelMemDims*KernelMemDims];
				atomicAdd((dImageMatrix+((xflr+a-1)*2)+((yflr+b-1)*ImageMatrixMemDims0*2)+((zflr+c-1)*ImageMatrixMemDims0*ImageMatrixMemDims1*2)),(KernVal*DatValR));
				atomicAdd((dImageMatrix+((xflr+a-1)*2+1)+((yflr+b-1)*ImageMatrixMemDims0*2)+((zflr+c-1)*ImageMatrixMemDims0*ImageMatrixMemDims1*2)),(KernVal*DatValI));
				}
			}
		}
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridComplex(size_t *GpuNum, size_t *HSampDat, size_t *HReconInfo, size_t *HKernel, size_t *HImageMatrix,
                            size_t *SampDatMemDims, size_t *KernelMemDims, size_t *ImageMatrixMemDims, 
                            size_t *iKern, size_t *KernHw, char *Error)
{
float *dSampDat,*dReconInfo,*dKernel,*dImageMatrix;
int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
int SampDatLength = SampDatMemDims[0]*SampDatMemDims[1];
int bpg = int(ceil(float(SampDatLength)/float(tpb)));  

hipSetDevice(GpuNum[0]);
dSampDat = (float*)HSampDat[GpuNum[0]];
dReconInfo = (float*)HReconInfo[GpuNum[0]];
dKernel = (float*)HKernel[GpuNum[0]];
dImageMatrix = (float*)HImageMatrix[GpuNum[0]];               
Conv3D<<<bpg,tpb>>>(dSampDat,dReconInfo,dKernel,dImageMatrix,KernelMemDims[0],ImageMatrixMemDims[0],ImageMatrixMemDims[1],iKern[0],KernHw[0],SampDatLength);

hipSetDevice(GpuNum[0]);

// cudaDeviceSynchronize();												// asynchronous exit
// const char* Error0 = cudaGetErrorString(cudaGetLastError());
// strcpy(Error,Error0);
}	
							
						
							