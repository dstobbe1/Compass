
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- Use CUDA's atomicAdd (only available on compute 6.x and later)
///  	- switch to lrint
///		(still to-do: remove (j<DatLen) check)
///==========================================================

extern "C" void ConvSamp2GridSplitRSM(int Count, size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);

					
///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDat, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
int xF,yF,zF;
int xflr,yflr,zflr;
float DatVal,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatVal = dSDat[j];
	xflr = __float2int_rd(dKx[j]);
    yflr = __float2int_rd(dKy[j]);
    zflr = __float2int_rd(dKz[j]);   
	//xF = lround(iKern*(dKx[j]-xflr));     		// halfway rounded away from zero      
    //yF = lround(iKern*(dKy[j]-yflr)); 
    //zF = lround(iKern*(dKz[j]-zflr));	
	xF = lrint(iKern*(dKx[j]-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(dKy[j]-yflr)); 
    zF = lrint(iKern*(dKz[j]-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}		
    for(c=-chW; c<=chW+1; c++) {
        for(b=-chW; b<=chW+1; b++) {
            for(a=-chW; a<=chW+1; a++) {
				KernVal = dKern[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernSz + lrint(fabsf(zF-(c*iKern)))*KernSz*KernSz];		
                atomicAdd((dGDat+(xflr+a-1)+((yflr+b-1)*GrdDatSz)+((zflr+c-1)*GrdDatSz*GrdDatSz)),(KernVal*DatVal)); 
				}
			}
		}  
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitRSM(int Count, size_t *HSampDat, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

    float *dSDat,*dGDat,*dKx,*dKy,*dKz,*dKern;
	int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
    int PtsPerDevice = int(ceil(float(DatLen)/float(Count)));
	int bpg = int(ceil(float(PtsPerDevice)/float(tpb)));  
    for (int n=0; n<Count; n++){	
        hipSetDevice(n);
        dSDat = (float*)(HSampDat[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));
        dGDat = (float*)HGrdDat[n];
        dKx = (float*)(HKx[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));
        dKy = (float*)(HKy[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));	
        dKz = (float*)(HKz[n]+(SampDatAdr+n*PtsPerDevice)*sizeof(float));
        dKern = (float*)HKern[n];	                
        Conv3D<<<bpg,tpb>>>(dSDat,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,PtsPerDevice,KernSz,iKern,chW);
    }
	hipDeviceSynchronize();												// make sure finished	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

