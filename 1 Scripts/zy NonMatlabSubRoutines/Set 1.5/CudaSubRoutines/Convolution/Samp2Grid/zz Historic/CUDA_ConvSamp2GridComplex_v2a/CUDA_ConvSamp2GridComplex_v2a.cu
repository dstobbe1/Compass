
#include <hip/hip_runtime.h>
///==========================================================
/// (v2a)
///		- 
///==========================================================

extern "C" void ConvSamp2GridComplex(size_t *GpuNum, size_t *HSampDat, size_t *HReconInfo, size_t *HKernel, size_t *HImageMatrix, size_t *ProjStart, 
                                        size_t *SampDatMemDims, size_t *ReconInfoMemDims, size_t *KernelMemDims, size_t *ImageMatrixMemDims, 
                                        size_t *iKern, size_t *KernHw, char *Error);
						

///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSampDat, float* dReconInfo, float* dKernel, float* dImageMatrix, int ProjStart,
					   int ReconInfoMemDims, int KernelMemDims0, int KernelMemDims1, int ImageMatrixMemDims0,  int ImageMatrixMemDims1, int iKern, int KernHw, int PtsPerDevice)
{
int Start;
float Kx,Ky,Kz,Sdc;
int xF,yF,zF;
int xflr,yflr,zflr;
float DatValR,DatValI;
float KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < PtsPerDevice) {

    Start = 4*ProjStart*ReconInfoMemDims;
    Kx = dReconInfo[Start+4*j];
    Ky = dReconInfo[Start+4*j+1];
    Kz = dReconInfo[Start+4*j+2];
    Sdc = dReconInfo[Start+4*j+3];

	xflr = __float2int_rd(Kx);
    yflr = __float2int_rd(Ky);
    zflr = __float2int_rd(Kz);   
	xF = lrint(iKern*(Kx-xflr)); 				// halfway rounded to nearest even integer (supposedly faster)     
    yF = lrint(iKern*(Ky-yflr)); 
    zF = lrint(iKern*(Kz-zflr));	
	if (xF == iKern){
        xflr = xflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        yflr = yflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        zflr = zflr + 1;
        zF = 0;
		}	

    DatValR = dSampDat[2*j]*Sdc;
    DatValI = dSampDat[2*j+1]*Sdc;	
    
    for(c=-KernHw; c<=KernHw+1; c++) {
        for(b=-KernHw; b<=KernHw+1; b++) {
            for(a=-KernHw; a<=KernHw+1; a++) {
				KernVal = dKernel[lrint(fabsf(xF-(a*iKern))) + lrint(fabsf(yF-(b*iKern)))*KernelMemDims0 + lrint(fabsf(zF-(c*iKern)))*KernelMemDims0*KernelMemDims1];		
				atomicAdd((dImageMatrix+((xflr+a-1)*2)+((yflr+b-1)*ImageMatrixMemDims0*2)+((zflr+c-1)*ImageMatrixMemDims0*ImageMatrixMemDims1*2)),(KernVal*DatValR));
				atomicAdd((dImageMatrix+((xflr+a-1)*2+1)+((yflr+b-1)*ImageMatrixMemDims0*2)+((zflr+c-1)*ImageMatrixMemDims0*ImageMatrixMemDims1*2)),(KernVal*DatValI));
				}
			}
		}
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridComplex(size_t *GpuNum, size_t *HSampDat, size_t *HReconInfo, size_t *HKernel, size_t *HImageMatrix, size_t *ProjStart, 
                            size_t *SampDatMemDims, size_t *ReconInfoMemDims, size_t *KernelMemDims, size_t *ImageMatrixMemDims, 
                            size_t *iKern, size_t *KernHw, char *Error)
{
float *dSampDat,*dReconInfo,*dKernel,*dImageMatrix;
int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1). Should be multiple of warp_size=32.
int PtsPerDevice = SampDatMemDims[0]*SampDatMemDims[1];
int bpg = int(ceil(float(PtsPerDevice)/float(tpb)));  
for (int n=0; n<GpuNum[0]; n++){	
    hipSetDevice(n);
    dSampDat = (float*)HSampDat[n];
    dReconInfo = (float*)HReconInfo[n];
    dKernel = (float*)HKernel[n];
    dImageMatrix = (float*)HImageMatrix[n];               
    Conv3D<<<bpg,tpb>>>(dSampDat,dReconInfo,dKernel,dImageMatrix,ProjStart[0],ReconInfoMemDims[0],KernelMemDims[0],KernelMemDims[1],ImageMatrixMemDims[0],ImageMatrixMemDims[1],iKern[0],KernHw[0],PtsPerDevice);
    }
hipDeviceSynchronize();												// make sure finished	
const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);
}	
							
						
							