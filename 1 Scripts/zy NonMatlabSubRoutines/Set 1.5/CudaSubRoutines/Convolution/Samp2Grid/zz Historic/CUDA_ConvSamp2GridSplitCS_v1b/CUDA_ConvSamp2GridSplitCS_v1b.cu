
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		- exactly same as (v1a) - name catch up with G2S
///==========================================================

extern "C" void ConvSamp2GridSplitCS(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error);


///=====================================================
/// Conv3D (kernel)					
///=====================================================
__global__ void Conv3D(float* dSDatR, float* dSDatI, float* dGDat, float* dKx, float* dKy, float* dKz, float* dKern, 
						int GrdDatSz, int DatLen, int KernSz, int iKern, int chW)
{
float xrmd,yrmd,zrmd;
float fxflr,fyflr,fzflr;
int xflr,yflr,zflr;
int xF,yF,zF;
float DatValR,DatValI,KernVal;
int j,a,b,c;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    DatValR = dSDatR[j];
	DatValI = dSDatI[j];
    fxflr = floorf(dKx[j]);
    fyflr = floorf(dKy[j]);
    fzflr = floorf(dKz[j]);
    xrmd = dKx[j]-fxflr;
    yrmd = dKy[j]-fyflr;
    zrmd = dKz[j]-fzflr;
    xF = lrintf(iKern*xrmd);                       
    yF = lrintf(iKern*yrmd); 
    zF = lrintf(iKern*zrmd);   
    if (xF == iKern){
        fxflr = fxflr + 1;
        xF = 0;
		}
    if (yF == iKern){
        fyflr = fyflr + 1;
        yF = 0;
		}
    if (zF == iKern){
        fzflr = fzflr + 1;
        zF = 0;
		}
	xflr = __float2int_rd(fxflr);
	yflr = __float2int_rd(fyflr);
	zflr = __float2int_rd(fzflr);	
    for(c=-chW; c<=chW+1; c++) {
        for(b=-chW; b<=chW+1; b++) {
            for(a=-chW; a<=chW+1; a++) {
                KernVal = dKern[lrintf(fabsf(xF-(a*iKern))+(fabsf(yF-(b*iKern))*KernSz)+(fabsf(zF-(c*iKern))*KernSz*KernSz))];
				atomicAdd((dGDat+((xflr+a-1)*2)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)),(KernVal*DatValR));
				atomicAdd((dGDat+((xflr+a-1)*2+1)+((yflr+b-1)*GrdDatSz*2)+((zflr+c-1)*GrdDatSz*GrdDatSz*2)),(KernVal*DatValI));
				}
			}
		}  
	}
}

///=====================================================
/// Code Entry
///=====================================================
void ConvSamp2GridSplitCS(size_t *HSampDatR, size_t *HSampDatI, size_t *HGrdDat, size_t *HKx, size_t *HKy, size_t *HKz, size_t *HKern, 
							int GrdDatSz, int DatLen, int KernSz, int iKern, int chW, int SampDatAdr, char* Error){

	float *dSDatR,*dSDatI,*dGDat,*dKx,*dKy,*dKz,*dKern;
	dSDatR = (float*)(*HSampDatR+SampDatAdr*sizeof(float));
	dSDatI = (float*)(*HSampDatI+SampDatAdr*sizeof(float));
	dGDat = (float*)*HGrdDat;
	dKx = (float*)(*HKx+SampDatAdr*sizeof(float));
	dKy = (float*)(*HKy+SampDatAdr*sizeof(float));	
	dKz = (float*)(*HKz+SampDatAdr*sizeof(float));
	dKern = (float*)*HKern;	
	
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(DatLen)/float(tpb)));                           
	Conv3D<<<bpg,tpb>>>(dSDatR,dSDatI,dGDat,dKx,dKy,dKz,dKern,GrdDatSz,DatLen,KernSz,iKern,chW);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	
	hipDeviceSynchronize();												// make sure finished	
}
							
