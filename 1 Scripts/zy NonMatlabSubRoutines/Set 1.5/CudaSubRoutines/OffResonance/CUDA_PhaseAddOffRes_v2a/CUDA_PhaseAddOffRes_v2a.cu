
#include <hip/hip_runtime.h>
///==========================================================
/// (v2a)
///		- input: real - imag (interleaved)
///==========================================================

extern "C" void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, float T, int ImLen, char *Error);

#define pi 3.141592

///==========================================================
/// PAOR (kernel)
///==========================================================
__global__ void PAOR(float *dIm0, float *dOff, float *dIm1, float T)
{	
int n = blockDim.x*blockIdx.x + threadIdx.x;
float val1 = dIm0[n];
float val2 = dIm0[n+1];
float val3 = dIm0[n-1];
float phaseaddreal = cosf(2*pi*T*dOff[n]);
float phaseaddimag = sinf(2*pi*T*dOff[n]);

if (n == 0) {
	dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
	}	
else if (n % 2) {
	dIm1[n] = val3*phaseaddimag + val1*phaseaddreal;
	}		
else {
	dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
	}
__syncthreads();
}

///==========================================================
/// PhaseAddOffRes
///==========================================================
void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, float T, int ImLen, char *Error){ 
	float *dIm0, *dOff, *dIm1;
	dIm0 = (float*)*HIm0;
	dOff = (float*)*HOff;
	dIm1 = (float*)*HIm1;	

	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(ImLen)/float(tpb)));                            
	PAOR<<<bpg,tpb>>>(dIm0,dOff,dIm1,T);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}