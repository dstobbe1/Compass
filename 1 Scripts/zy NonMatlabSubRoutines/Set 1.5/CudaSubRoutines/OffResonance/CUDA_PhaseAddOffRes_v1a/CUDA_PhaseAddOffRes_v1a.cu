
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- 
///==========================================================

extern "C" void PhaseAddOffRes(size_t *HIm0M, size_t *HIm0P, size_t *HOff, size_t *HIm, float T, int ImLen, char *Error);

#define pi 3.141592

///==========================================================
/// PAOR (kernel)
///==========================================================
__global__ void PAOR(float *dIm0M, float *dIm0P, float *dOff, float *dIm, float T)
{	
int n;
n = blockDim.x*blockIdx.x + threadIdx.x;
if (n == 0) {
	dIm[n] = dIm0M[n]*cosf(2*pi*T*dOff[n] + dIm0P[n]);	
	}	
else if (n % 2) {
	dIm[n] = dIm0M[n]*sinf(2*pi*T*dOff[n] + dIm0P[n]);
	}		
else {
	dIm[n] = dIm0M[n]*cosf(2*pi*T*dOff[n] + dIm0P[n]);	
	}
__syncthreads();
}

///==========================================================
/// PhaseAddOffRes
///==========================================================
void PhaseAddOffRes(size_t *HIm0M, size_t *HIm0P, size_t *HOff, size_t *HIm, float T, int ImLen, char *Error){ 
	float *dIm0M, *dIm0P, *dOff, *dIm;
	dIm0M = (float*)*HIm0M;
	dIm0P = (float*)*HIm0P;
	dOff = (float*)*HOff;
	dIm = (float*)*HIm;	

	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(float(ImLen)/float(tpb)));                            
	PAOR<<<bpg,tpb>>>(dIm0M,dIm0P,dOff,dIm,T);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}