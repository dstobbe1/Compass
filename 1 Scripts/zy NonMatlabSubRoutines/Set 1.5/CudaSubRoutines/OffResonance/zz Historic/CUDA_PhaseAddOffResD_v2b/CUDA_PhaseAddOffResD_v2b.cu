
#include <hip/hip_runtime.h>
///==========================================================
/// (v2b)
///		- boundary on 'n' so doesn't overwrite
///		- '+1' and '-1' accesses to appropriate places (no invalid access)
///		- add 'cudaDeviceSynchronize()' - to know error comes from here
///==========================================================

extern "C" void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, double T, int ImLen, char *Error);

#define pi 3.141592

///==========================================================
/// PAOR (kernel)
///==========================================================
__global__ void PAOR(double *dIm0, double *dOff, double *dIm1, double T, int ImLen)
{	
int n = blockDim.x*blockIdx.x + threadIdx.x;
if (n < ImLen) {
	double val1 = dIm0[n];
	double phaseaddreal = cos(2*pi*T*dOff[n]);
	double phaseaddimag = sin(2*pi*T*dOff[n]);
	if (n == 0) {
		double val2 = dIm0[n+1];
		dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
		}	
	else if (n % 2) {											// i.e. n = odd
		double val3 = dIm0[n-1];
		dIm1[n] = val3*phaseaddimag + val1*phaseaddreal;
		}		
	else {
		double val2 = dIm0[n+1];
		dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
		}
	}
__syncthreads();	
}

///==========================================================
/// PhaseAddOffRes
///==========================================================
void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, double T, int ImLen, char *Error){ 
	double *dIm0, *dOff, *dIm1;
	dIm0 = (double*)*HIm0;
	dOff = (double*)*HOff;
	dIm1 = (double*)*HIm1;	

	int tpb = 1024;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil((double(ImLen)/double(tpb))-0.000001));                            
	PAOR<<<bpg,tpb>>>(dIm0,dOff,dIm1,T,ImLen);

	hipDeviceSynchronize();												// make sure finished		
	
	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

