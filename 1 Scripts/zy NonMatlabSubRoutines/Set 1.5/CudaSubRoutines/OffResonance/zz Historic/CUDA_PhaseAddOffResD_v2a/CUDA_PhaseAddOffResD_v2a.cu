
#include <hip/hip_runtime.h>
///==========================================================
/// (v2a)
///		- input: real - imag (interleaved)
///		'D' is doubles
///==========================================================

extern "C" void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, double T, int ImLen, char *Error);

#define pi 3.141592

///==========================================================
/// PAOR (kernel)
///==========================================================
__global__ void PAOR(double *dIm0, double *dOff, double *dIm1, double T)
{	
int n = blockDim.x*blockIdx.x + threadIdx.x;
double val1 = dIm0[n];
double val2 = dIm0[n+1];
double val3 = dIm0[n-1];
double phaseaddreal = cos(2*pi*T*dOff[n]);
double phaseaddimag = sin(2*pi*T*dOff[n]);

if (n == 0) {
	dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
	}	
else if (n % 2) {
	dIm1[n] = val3*phaseaddimag + val1*phaseaddreal;
	}		
else {
	dIm1[n] = val1*phaseaddreal - val2*phaseaddimag;	
	}
__syncthreads();
}

///==========================================================
/// PhaseAddOffRes
///==========================================================
void PhaseAddOffRes(size_t *HIm0, size_t *HOff, size_t *HIm1, double T, int ImLen, char *Error){ 
	double *dIm0, *dOff, *dIm1;
	dIm0 = (double*)*HIm0;
	dOff = (double*)*HOff;
	dIm1 = (double*)*HIm1;	

	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(ImLen)/double(tpb)));                            
	PAOR<<<bpg,tpb>>>(dIm0,dOff,dIm1,T);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}