
#include <hip/hip_runtime.h>
///==========================================================
/// (v1d)
///		- Recompile with CUDA 7.5
///==========================================================

extern "C" void FFTShift(size_t *HDatC0, size_t *HDatC1, int MatSz, char *Error);

#define BLKDIM 8
#define pi 3.141592

///==========================================================
/// FFTSh (kernel)
///==========================================================
__global__ void FFTSh(double *dDatC0, double *dDatC1, int MatSz)
{	
int nx = blockDim.x*blockIdx.x + threadIdx.x;
int ny = blockDim.y*blockIdx.y + threadIdx.y;
int nz = blockDim.z*blockIdx.z + threadIdx.z;

if ((nx<MatSz*2) && (ny<MatSz) && (nz<MatSz)) {
	int kx = nx+MatSz;
	if (kx>=(MatSz*2)){
		kx = kx-(MatSz*2);
		}
	int ky = ny+MatSz/2;
	if (ky>=MatSz){
		ky = ky-MatSz;
		}
	int kz = nz+MatSz/2;
	if (kz>=MatSz){
		kz = kz-MatSz;
		}		
	dDatC1[(kx)+(ky*MatSz*2)+(kz*MatSz*MatSz*2)] = dDatC0[(nx)+(ny*MatSz*2)+(nz*MatSz*MatSz*2)];
	}
__syncthreads();
}

///==========================================================
/// FFTShift
///==========================================================
void FFTShift(size_t *HDatC0, size_t *HDatC1, int MatSz, char *Error){ 
	double *dDatC0,*dDatC1;
	dDatC0 = (double*)*HDatC0;
	dDatC1 = (double*)*HDatC1;
	
	dim3 tpbmat(BLKDIM*2,BLKDIM,BLKDIM);  									// possible to go up to 1024 (16x8x8=1024). Should be multiple of warp_size=32.
	int bpg = int(ceil(double(MatSz)/double(BLKDIM)));  
	dim3 bpgmat(bpg,bpg,bpg);
	                         
	FFTSh<<<bpgmat,tpbmat>>>(dDatC0,dDatC1,MatSz);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}