
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		use CUDAs 3D addressing
///==========================================================

extern "C" void FFTShift(size_t *HkDatC0, size_t *HkDatC1, int ImSz, char *Error);

#define BLKDIM 8
#define pi 3.141592

///==========================================================
/// FFTSh (kernel)
///==========================================================
__global__ void FFTSh(double *dkDatC0, double *dkDatC1, int ImSz)
{	
int nx = blockDim.x*blockIdx.x + threadIdx.x;
int ny = blockDim.y*blockIdx.y + threadIdx.y;
int nz = blockDim.z*blockIdx.z + threadIdx.z;
int kx = nx+ImSz/2;
if (kx>ImSz){
	kx = kx-ImSz;
	}
int ky = ny+ImSz/2;
if (ky>ImSz){
	ky = ky-ImSz;
	}
int kz = nz+ImSz/2;
if (kz>ImSz){
	kz = kz-ImSz;
	}	
dkDatC1[(kx)+(ky*ImSz)+(kz*ImSz*ImSz)] = dkDatC0[(nx)+(ny*ImSz)+(nz*ImSz*ImSz)];
__syncthreads();
}

///==========================================================
/// FFTShift
///==========================================================
void FFTShift(size_t *HkDatC0, size_t *HkDatC1, int ImSz, char *Error){ 
	double *dkDatC0,*dkDatC1;
	dkDatC0 = (double*)*HkDatC0;
	dkDatC1 = (double*)*HkDatC1;
	
	dim3 tpbmat(BLKDIM,BLKDIM,BLKDIM);  									// possible to go up to 1024 (8x8x8=512). Should be multiple of warp_size=32.
	int bpg = int(ceil(double(ImSz)/double(BLKDIM)));  
	dim3 bpgmat(bpg,bpg,bpg);
	                         
	FFTSh<<<bpgmat,tpbmat>>>(dkDatC0,dkDatC1,ImSz);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}