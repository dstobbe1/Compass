#include "hip/hip_runtime.h"
///==========================================================
/// (v1a)
///		'D' is doubles
///==========================================================

extern "C" void FFTShift(size_t *HkDatC, int ImSz, char *Error);

#define pi 3.141592

///==========================================================
/// FFTS1 (kernel)
///==========================================================
__global__ void FFTS1(double *dkDatC, int ImSz, int ImLen, double T)
{	
int n = blockDim.x*blockIdx.x + threadIdx.x;
int k = n+ImLen/2;
if (k>ImLen){
	k = k-ImLen;
	}
dkDatC[k] = dkDatC[n];
__syncthreads();
}

///==========================================================
/// FFTS2 (kernel)
///==========================================================
__global__ void FFTS2(double *dkDatC, int ImSz, int ImLen, double T)
{	
int n = blockDim.x*blockIdx.x + threadIdx.x;
int k = n+ImLen/2;
if (k>ImLen){
	k = k-ImLen;
	}
dkDatC[k] = dkDatC[n];
__syncthreads();
}

///==========================================================
/// PhaseAddOffRes
///==========================================================
void FFTShift(size_t *HkDatC, int ImSz, char *Error){ 
	double *dkDatC;
	dkDatC = (double*)*HIm0;
	int ImLen = ImSz*ImSz*Imsz;
	int tpb = 512;                                                          // possible to go up to 1024. Should be multiple of warp_size=32.
	int bpg = int(ceil(double(ImLen)/double(tpb)));                            
	FFTS1<<<bpg,tpb>>>(dkDatC,ImSz,ImLen);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}