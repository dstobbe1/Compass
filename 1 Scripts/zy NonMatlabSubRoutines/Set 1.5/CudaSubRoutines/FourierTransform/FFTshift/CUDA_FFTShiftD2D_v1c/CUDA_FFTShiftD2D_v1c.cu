
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		use CUDAs 3D addressing
///==========================================================

extern "C" void FFTShift2D(size_t *HDatC0, size_t *HDatC1, int MatSz, char *Error);

#define BLKDIM 16
#define pi 3.141592

///==========================================================
/// FFTSh (kernel)
///==========================================================
__global__ void FFTSh(double *dDatC0, double *dDatC1, int MatSz)
{	
int nx = blockDim.x*blockIdx.x + threadIdx.x;
int ny = blockDim.y*blockIdx.y + threadIdx.y;

if ((nx<MatSz*2) && (ny<MatSz)) {
	int kx = nx+MatSz;
	if (kx>=(MatSz*2)){
		kx = kx-(MatSz*2);
		}
	int ky = ny+MatSz/2;
	if (ky>=MatSz){
		ky = ky-MatSz;
		}	
	dDatC1[(kx)+(ky*MatSz*2)] = dDatC0[(nx)+(ny*MatSz*2)];
	}
__syncthreads();
}

///==========================================================
/// FFTShift
///==========================================================
void FFTShift2D(size_t *HDatC0, size_t *HDatC1, int MatSz, char *Error){ 
	double *dDatC0,*dDatC1;
	dDatC0 = (double*)*HDatC0;
	dDatC1 = (double*)*HDatC1;
	
	dim3 tpbmat(BLKDIM*2,BLKDIM);  									// possible to go up to 1024 (32x16=512). Should be multiple of warp_size=32.
	int bpg = int(ceil(double(MatSz)/double(BLKDIM)));  
	dim3 bpgmat(bpg,bpg);
	                         
	FFTSh<<<bpgmat,tpbmat>>>(dDatC0,dDatC1,MatSz);

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}