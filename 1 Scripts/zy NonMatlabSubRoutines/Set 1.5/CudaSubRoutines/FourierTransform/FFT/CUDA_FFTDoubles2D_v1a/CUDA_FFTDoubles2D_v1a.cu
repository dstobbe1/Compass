///==========================================================
/// (v1b)
///		- Remove Matrix loading/unloading from 'Setup' and 'Teardown'
///==========================================================

#include "hipfft/hipfft.h"

extern "C" void FFT2Dsetup(hipfftHandle *plan, int MatSz, char *Error);
extern "C" void FFT2D(size_t *HdIm, size_t *HdkDat, hipfftHandle *plan, char *Error);
extern "C" void FFT2Dteardown(hipfftHandle *plan);

///==========================================================
/// FFT2Dsetup
///==========================================================
void FFT2Dsetup(hipfftHandle *plan, int MatSz, char *Error){
	const char* Error0; 
	hipfftPlan2d(plan,MatSz,MatSz,HIPFFT_Z2Z);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// FFT2D
///==========================================================
void FFT2D(size_t *HdIm, size_t *HdkDat, hipfftHandle *plan, char *Error){
	const char* Error0; 
	hipfftDoubleComplex *dIm, *dkDat;
	dIm = (hipfftDoubleComplex*)*HdIm;
	dkDat = (hipfftDoubleComplex*)*HdkDat;
	hipfftExecZ2Z(*plan,dIm,dkDat,HIPFFT_FORWARD);
	if (hipGetLastError() != hipSuccess){
		Error0 = hipGetErrorString(hipGetLastError());
		strcpy(Error,Error0); return;
	}
	hipDeviceSynchronize();									// test if needed...
	if (hipGetLastError() != hipSuccess){
		Error0 = hipGetErrorString(hipGetLastError());
		strcpy(Error,Error0); return;
	}
}

///==========================================================
/// FFT2Dteardown
///==========================================================
void FFT2Dteardown(hipfftHandle *plan){
	hipfftDestroy(*plan);
}


