///==========================================================
/// (v1b)
///		- Remove Matrix loading/unloading from 'Setup' and 'Teardown'
///==========================================================

#include "hipfft/hipfft.h"

extern "C" void FFT3Dsetup(hipfftHandle *plan, int MatSz, char *Error);
extern "C" void FFT3D(size_t *HdIm, size_t *HdkDat, hipfftHandle *plan, char *Error);
extern "C" void FFT3Dteardown(hipfftHandle *plan);

///==========================================================
/// FFT3Dsetup
///==========================================================
void FFT3Dsetup(hipfftHandle *plan, int MatSz, char *Error){
	const char* Error0; 
	hipfftPlan3d(plan,MatSz,MatSz,MatSz,HIPFFT_Z2Z);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// FFT3D
///==========================================================
void FFT3D(size_t *HdIm, size_t *HdkDat, hipfftHandle *plan, char *Error){
	const char* Error0; 
	hipfftDoubleComplex *dIm, *dkDat;
	dIm = (hipfftDoubleComplex*)*HdIm;
	dkDat = (hipfftDoubleComplex*)*HdkDat;
	hipfftExecZ2Z(*plan,dIm,dkDat,HIPFFT_FORWARD);
	if (hipGetLastError() != hipSuccess){
		Error0 = hipGetErrorString(hipGetLastError());
		strcpy(Error,Error0); return;
	}
	hipDeviceSynchronize();									// test if needed...
	if (hipGetLastError() != hipSuccess){
		Error0 = hipGetErrorString(hipGetLastError());
		strcpy(Error,Error0); return;
	}
}

///==========================================================
/// FFT3Dteardown
///==========================================================
void FFT3Dteardown(hipfftHandle *plan){
	hipfftDestroy(*plan);
}


