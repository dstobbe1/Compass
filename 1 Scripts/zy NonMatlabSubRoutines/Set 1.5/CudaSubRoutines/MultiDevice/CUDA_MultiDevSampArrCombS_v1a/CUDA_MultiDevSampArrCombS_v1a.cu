
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void MultiDevSampArrCombS(int Dst, int Src, size_t *HSampDat, size_t *HSampDatTemp, int DatLen, char* Error);
						

///=====================================================
/// SampArrCombS (kernel)					
///=====================================================
__global__ void SampArrCombS(float* dSampDatDst, float* dSampDatDstTemp, int DatLen)
{
int j;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    dSampDatDstTemp[j] = dSampDatDstTemp[j] + dSampDatDst[j];
	}
}

///=====================================================
/// Code Entry
///=====================================================
void MultiDevSampArrCombS(int Dst, int Src, size_t *HSampDat, size_t *HSampDatTemp, int DatLen, char* Error){
    
    float *dSampDatDst,*dSampDatSrc,*dSampDatDstTemp;
    dSampDatDst = (float*)HSampDat[Dst];
    dSampDatSrc = (float*)HSampDat[Src];
    dSampDatDstTemp = (float*)HSampDatTemp[Dst];
    size_t DatLenBytes = sizeof(float)*DatLen;

    hipMemcpyPeer(dSampDatDstTemp,Dst,dSampDatSrc,Src,DatLenBytes);
	hipDeviceSynchronize();												// make sure finished	

    int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1)
	int bpg = int(ceil(float(DatLen)/float(tpb)));  
    hipSetDevice(Dst);
    SampArrCombS<<<bpg,tpb>>>(dSampDatDst,dSampDatDstTemp,DatLen);
	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

