
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void MultiDevMatAdd(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error);
						

///=====================================================
/// MatAdd (kernel)					
///=====================================================
__global__ void MatAdd(double* dGdatDst, double* dGdatDstTemp, int MatLen)
{
int j;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < MatLen) {
    dGdatDstTemp[j] = dGdatDstTemp[j] + dGdatDst[j];
	}
}

///=====================================================
/// Code Entry
///=====================================================
void MultiDevMatAdd(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error){
    
    double *dGdatDst,*dGdatSrc,*dGdatDstTemp;
    dGdatDst = (double*)HGrdDat[Dst];
    dGdatSrc = (double*)HGrdDat[Src];
    dGdatDstTemp = (double*)HGrdDatTemp[Dst];
    int MatLen = MatSize*MatSize*MatSize;
    size_t MatSizeBytes = sizeof(double)*MatLen;

    hipMemcpyPeer(dGdatDstTemp,Dst,dGdatSrc,Src,MatSizeBytes);
	hipDeviceSynchronize();												// make sure finished	

    int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1)
	int bpg = int(ceil(double(MatLen)/double(tpb)));  
    hipSetDevice(Dst);
    MatAdd<<<bpg,tpb>>>(dGdatDst,dGdatDstTemp,MatLen);
	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

