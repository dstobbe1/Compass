
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void MultiDevSampArrComb(int Dst, int Src, size_t *HSampDat, size_t *HSampDatTemp, int DatLen, char* Error);
						

///=====================================================
/// SampArrComb (kernel)					
///=====================================================
__global__ void SampArrComb(double* dSampDatDst, double* dSampDatDstTemp, int DatLen)
{
int j;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < DatLen) {
    dSampDatDstTemp[j] = dSampDatDstTemp[j] + dSampDatDst[j];
	}
}

///=====================================================
/// Code Entry
///=====================================================
void MultiDevSampArrComb(int Dst, int Src, size_t *HSampDat, size_t *HSampDatTemp, int DatLen, char* Error){
    
    double *dSampDatDst,*dSampDatSrc,*dSampDatDstTemp;
    dSampDatDst = (double*)HSampDat[Dst];
    dSampDatSrc = (double*)HSampDat[Src];
    dSampDatDstTemp = (double*)HSampDatTemp[Dst];
    size_t DatLenBytes = sizeof(double)*DatLen;

    hipMemcpyPeer(dSampDatDstTemp,Dst,dSampDatSrc,Src,DatLenBytes);
	hipDeviceSynchronize();												// make sure finished	

    int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1)
	int bpg = int(ceil(double(DatLen)/double(tpb)));  
    hipSetDevice(Dst);
    SampArrComb<<<bpg,tpb>>>(dSampDatDst,dSampDatDstTemp,DatLen);
	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

