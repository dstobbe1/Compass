
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void MultiDevMatAddCS(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error);
						

///=====================================================
/// MatAdd (kernel)					
///=====================================================
__global__ void MatAdd(float* dGdatDst, float* dGdatDstTemp, int MatLen)
{
int j;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < MatLen) {
    dGdatDstTemp[j] = dGdatDstTemp[j] + dGdatDst[j];
	}
}

///=====================================================
/// Code Entry
///=====================================================
void MultiDevMatAddCS(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error){
    
    float *dGdatDst,*dGdatSrc,*dGdatDstTemp;
    dGdatDst = (float*)HGrdDat[Dst];
    dGdatSrc = (float*)HGrdDat[Src];
    dGdatDstTemp = (float*)HGrdDatTemp[Dst];
    int MatLen = MatSize*MatSize*MatSize*2;
    size_t MatSizeBytes = sizeof(float)*MatLen;

    hipMemcpyPeer(dGdatDstTemp,Dst,dGdatSrc,Src,MatSizeBytes);
	hipDeviceSynchronize();												// make sure finished	

    int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1)
	int bpg = int(ceil(float(MatLen)/float(tpb)));  
    hipSetDevice(Dst);
    MatAdd<<<bpg,tpb>>>(dGdatDst,dGdatDstTemp,MatLen);
	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

