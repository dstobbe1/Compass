
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		
///==========================================================

extern "C" void MultiDevMatAddS(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error);
						

///=====================================================
/// MatAddS (kernel)					
///=====================================================
__global__ void MatAddS(float* dGdatDst, float* dGdatDstTemp, int MatLen)
{
int j;
j = blockDim.x*blockIdx.x + threadIdx.x;

if (j < MatLen) {
    dGdatDstTemp[j] = dGdatDstTemp[j] + dGdatDst[j];
	}
}

///=====================================================
/// Code Entry
///=====================================================
void MultiDevMatAddS(int Dst, int Src, size_t *HGrdDat, size_t *HGrdDatTemp, int MatSize, char* Error){
    
    float *dGdatDst,*dGdatSrc,*dGdatDstTemp;
    dGdatDst = (float*)HGrdDat[Dst];
    dGdatSrc = (float*)HGrdDat[Src];
    dGdatDstTemp = (float*)HGrdDatTemp[Dst];
    int MatLen = MatSize*MatSize*MatSize;
    size_t MatSizeBytes = sizeof(float)*MatLen;

    hipMemcpyPeer(dGdatDstTemp,Dst,dGdatSrc,Src,MatSizeBytes);
	hipDeviceSynchronize();												// make sure finished	

    int tpb = 128;                                                          // Equal to number of cores in multiprocessor (6.1)
	int bpg = int(ceil(float(MatLen)/float(tpb)));  
    hipSetDevice(Dst);
    MatAddS<<<bpg,tpb>>>(dGdatDst,dGdatDstTemp,MatLen);
	hipDeviceSynchronize();												// make sure finished	

	const char* Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
							

