
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- Recompile with CUDA 9.0 (compute 6.1)
///==========================================================

extern "C" void CUDAcount(int *Count, char *Error);
//extern "C" void CUDAselect(int device, char *Error);
//extern "C" void CUDAreset(char *Error);

extern "C" void ArrAllocSgl(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrAllocSglC(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrInitSgl(int Count, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrInitSglC(int Count, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrFreeSgl(int Count, size_t *HMat, char *Error);
extern "C" void ArrFreeSglC(int Count, size_t *HMat, char *Error);
extern "C" void ArrLoadSgl(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrLoadSglC(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnSgl(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnSglC(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error);

extern "C" void Mat3DAllocSgl(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DAllocSglC(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DInitSgl(int Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitSglC(int Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DFreeSgl(int Count,size_t *HMat, char *Error);
extern "C" void Mat3DFreeSglC(int Count, size_t *HMat, char *Error);
extern "C" void Mat3DLoadSgl(int Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DLoadSglC(int Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSgl(int Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSglC(int Count, float *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAcount(int *Count, char *Error){ 
	const char* Error0; 
	hipGetDeviceCount(Count);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocSgl
///==========================================================
void ArrAllocSgl(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocSglC
///==========================================================
void ArrAllocSglC(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSgl
///==========================================================
void ArrInitSgl(int Count, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglC
///==========================================================
void ArrInitSglC(int Count, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSgl
///==========================================================
void ArrFreeSgl(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglC
///==========================================================
void ArrFreeSglC(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSgl
///==========================================================
void ArrLoadSgl(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglC
///==========================================================
void ArrLoadSglC(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSgl
///==========================================================
void ArrReturnSgl(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	
    hipSetDevice(Count);
    dMat = (float*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglC
///==========================================================
void ArrReturnSglC(int Count, float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	
    hipSetDevice(Count);
    dMat = (float*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat3DAllocSgl
///==========================================================
void Mat3DAllocSgl(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DAllocSglC
///==========================================================
void Mat3DAllocSglC(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSgl
///==========================================================
void Mat3DInitSgl(int Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSglC
///==========================================================
void Mat3DInitSglC(int Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSgl
///==========================================================
void Mat3DFreeSgl(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSglC
///==========================================================
void Mat3DFreeSglC(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSgl
///==========================================================
void Mat3DLoadSgl(int Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSglC
///==========================================================
void Mat3DLoadSglC(int Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSgl
///==========================================================
void Mat3DReturnSgl(int Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;

    hipSetDevice(Count);
    dMat = (float*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSglC
///==========================================================
void Mat3DReturnSglC(int Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;

    hipSetDevice(Count);
    dMat = (float*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
