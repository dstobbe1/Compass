
#include <hip/hip_runtime.h>
///==========================================================
/// (v1f)
///		- Drop SampDat
///     - Simplify 'ArrLoadSglOne'
///==========================================================

extern "C" void CUDAcount(int *Count, char *Error);
extern "C" void ArrAllocSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrAllocSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrFreeSglAll(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrFreeSglAllC(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrLoadSglAll(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAcount(int *Count, char *Error){ 
	const char* Error0; 
	hipGetDeviceCount(Count);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocSglAll
///==========================================================
void ArrAllocSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
    const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();
}

///==========================================================
/// ArrAllocSglAllC
///==========================================================
void ArrAllocSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAll
///==========================================================
void ArrInitSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAllC
///==========================================================
void ArrInitSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSglAll
///==========================================================
void ArrFreeSglAll(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglAllC
///==========================================================
void ArrFreeSglAllC(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAll
///==========================================================
void ArrLoadSglAll(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAllC
///==========================================================
void ArrLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrLoadSglOne
///==========================================================
void ArrLoadSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglOneC
///==========================================================
void ArrLoadSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSglOne
///==========================================================
void ArrReturnSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglOneC
///==========================================================
void ArrReturnSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}



