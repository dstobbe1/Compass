
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- Recompile with CUDA 9.0 (compute 6.1)
///==========================================================

extern "C" void CUDAcount(int *Count, char *Error);
//extern "C" void CUDAselect(int device, char *Error);
//extern "C" void CUDAreset(char *Error);

extern "C" void ArrAllocDbl(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrAllocDblC(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrInitDbl(int Count, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrInitDblC(int Count, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrFreeDbl(int Count, size_t *HMat, char *Error);
extern "C" void ArrFreeDblC(int Count, size_t *HMat, char *Error);
extern "C" void ArrLoadDbl(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrLoadDblC(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDbl(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDblC(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error);

extern "C" void Mat3DAllocDbl(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DAllocDblC(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DInitDbl(int Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitDblC(int Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DFreeDbl(int Count,size_t *HMat, char *Error);
extern "C" void Mat3DFreeDblC(int Count, size_t *HMat, char *Error);
extern "C" void Mat3DLoadDbl(int Count, double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DLoadDblC(int Count, double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnDbl(int Count, double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnDblC(int Count, double *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAcount(int *Count, char *Error){ 
	const char* Error0; 
	hipGetDeviceCount(Count);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocDbl
///==========================================================
void ArrAllocDbl(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocDblC
///==========================================================
void ArrAllocDblC(int Count, size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDbl
///==========================================================
void ArrInitDbl(int Count, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDblC
///==========================================================
void ArrInitDblC(int Count, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeDbl
///==========================================================
void ArrFreeDbl(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeDblC
///==========================================================
void ArrFreeDblC(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDbl
///==========================================================
void ArrLoadDbl(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDblC
///==========================================================
void ArrLoadDblC(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDbl
///==========================================================
void ArrReturnDbl(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	
    hipSetDevice(Count);
    dMat = (double*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnDblC
///==========================================================
void ArrReturnDblC(int Count, double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	
    hipSetDevice(Count);
    dMat = (double*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat3DAllocDbl
///==========================================================
void Mat3DAllocDbl(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DAllocDblC
///==========================================================
void Mat3DAllocDblC(int Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitDbl
///==========================================================
void Mat3DInitDbl(int Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitDblC
///==========================================================
void Mat3DInitDblC(int Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeDbl
///==========================================================
void Mat3DFreeDbl(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeDblC
///==========================================================
void Mat3DFreeDblC(int Count, size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadDbl
///==========================================================
void Mat3DLoadDbl(int Count, double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadDblC
///==========================================================
void Mat3DLoadDblC(int Count, double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count;n++){
		hipSetDevice(n);
		dMat = (double*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnDbl
///==========================================================
void Mat3DReturnDbl(int Count, double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;

    hipSetDevice(Count);
    dMat = (double*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnDblC
///==========================================================
void Mat3DReturnDblC(int Count, double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;

    hipSetDevice(Count);
    dMat = (double*)HMat[Count];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
