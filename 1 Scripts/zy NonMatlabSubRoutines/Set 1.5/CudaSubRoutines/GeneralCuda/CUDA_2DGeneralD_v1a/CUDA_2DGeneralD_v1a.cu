
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- 
///==========================================================

extern "C" void ArrAllocDbl(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrAllocDblC(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrInitDbl(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrInitDblC(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrFreeDbl(size_t *HMat, char *Error);
extern "C" void ArrFreeDblC(size_t *HMat, char *Error);
extern "C" void ArrLoadDbl(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrLoadDblC(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDbl(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDblC(double *Mat, size_t *HMat, int ArrSz, char *Error);

extern "C" void Mat2DAllocDbl(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat2DAllocDblC(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat2DInitDbl(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat2DInitDblC(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat2DFreeDbl(size_t *HMat, char *Error);
extern "C" void Mat2DFreeDblC(size_t *HMat, char *Error);
extern "C" void Mat2DLoadDbl(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat2DLoadDblC(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat2DReturnDbl(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat2DReturnDblC(double *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// ArrAllocDbl
///==========================================================
void ArrAllocDbl(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocDblC
///==========================================================
void ArrAllocDblC(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDbl
///==========================================================
void ArrInitDbl(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDblC
///==========================================================
void ArrInitDblC(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeDbl
///==========================================================
void ArrFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeDblC
///==========================================================
void ArrFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDbl
///==========================================================
void ArrLoadDbl(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDblC
///==========================================================
void ArrLoadDblC(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDbl
///==========================================================
void ArrReturnDbl(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDblC
///==========================================================
void ArrReturnDblC(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat2DAllocDbl
///==========================================================
void Mat2DAllocDbl(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DAllocDblC
///==========================================================
void Mat2DAllocDblC(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DInitDbl
///==========================================================
void Mat2DInitDbl(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DInitDblC
///==========================================================
void Mat2DInitDblC(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DFreeDbl
///==========================================================
void Mat2DFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DFreeDblC
///==========================================================
void Mat2DFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DLoadDbl
///==========================================================
void Mat2DLoadDbl(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DLoadDblC
///==========================================================
void Mat2DLoadDblC(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DReturnDbl
///==========================================================
void Mat2DReturnDbl(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DReturnDblC
///==========================================================
void Mat2DReturnDblC(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
