
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- Recompile with CUDA 9.0 (compute 6.1)
///==========================================================

extern "C" void CUDAselect(int device, char *Error);
extern "C" void CUDAreset(char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAselect(int device, char *Error){ 
	const char* Error0; 
	hipSetDevice(device);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// CUDA reset
///==========================================================
void CUDAreset(char *Error){ 
	const char* Error0; 
	hipDeviceReset();
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
