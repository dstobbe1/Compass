
#include <hip/hip_runtime.h>
///==========================================================
/// (v1c)
///		- Recompile with CUDA 9.0
///==========================================================

extern "C" void ArrAllocDbl(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrAllocDblC(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrInitDbl(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrInitDblC(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrFreeDbl(size_t *HMat, char *Error);
extern "C" void ArrFreeDblC(size_t *HMat, char *Error);
extern "C" void ArrLoadDbl(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrLoadDblC(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDbl(double *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnDblC(double *Mat, size_t *HMat, int ArrSz, char *Error);

extern "C" void Mat3DAllocDbl(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DAllocDblC(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DInitDbl(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitDblC(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DFreeDbl(size_t *HMat, char *Error);
extern "C" void Mat3DFreeDblC(size_t *HMat, char *Error);
extern "C" void Mat3DLoadDbl(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DLoadDblC(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnDbl(double *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnDblC(double *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// ArrAllocDbl
///==========================================================
void ArrAllocDbl(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocDblC
///==========================================================
void ArrAllocDblC(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDbl
///==========================================================
void ArrInitDbl(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDblC
///==========================================================
void ArrInitDblC(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeDbl
///==========================================================
void ArrFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeDblC
///==========================================================
void ArrFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDbl
///==========================================================
void ArrLoadDbl(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDblC
///==========================================================
void ArrLoadDblC(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDbl
///==========================================================
void ArrReturnDbl(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDblC
///==========================================================
void ArrReturnDblC(double *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat3DAllocDbl
///==========================================================
void Mat3DAllocDbl(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DAllocDblC
///==========================================================
void Mat3DAllocDblC(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitDbl
///==========================================================
void Mat3DInitDbl(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitDblC
///==========================================================
void Mat3DInitDblC(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeDbl
///==========================================================
void Mat3DFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeDblC
///==========================================================
void Mat3DFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadDbl
///==========================================================
void Mat3DLoadDbl(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadDblC
///==========================================================
void Mat3DLoadDblC(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnDbl
///==========================================================
void Mat3DReturnDbl(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnDblC
///==========================================================
void Mat3DReturnDblC(double *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*MatSz*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
