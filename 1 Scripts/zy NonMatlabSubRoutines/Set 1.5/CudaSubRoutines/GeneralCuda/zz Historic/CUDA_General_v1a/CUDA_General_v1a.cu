
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- 
///==========================================================

extern "C" void ArrAllocSgl(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrAllocSglC(size_t *HMat, int ArrSz, size_t *Tst, char *Error);
extern "C" void ArrInitSgl(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrInitSglC(size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrFreeSgl(size_t *HMat, char *Error);
extern "C" void ArrFreeSglC(size_t *HMat, char *Error);
extern "C" void ArrLoadSgl(float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrLoadSglC(float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnSgl(float *Mat, size_t *HMat, int ArrSz, char *Error);
extern "C" void ArrReturnSglC(float *Mat, size_t *HMat, int ArrSz, char *Error);

extern "C" void Mat3DAllocSgl(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DAllocSglC(size_t *HMat, int MatSz, size_t *Tst, char *Error);
extern "C" void Mat3DInitSgl(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitSglC(size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DFreeSgl(size_t *HMat, char *Error);
extern "C" void Mat3DFreeSglC(size_t *HMat, char *Error);
extern "C" void Mat3DLoadSgl(float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DLoadSglC(float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSgl(float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSglC(float *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// ArrAllocSgl
///==========================================================
void ArrAllocSgl(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*ArrSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocSglC
///==========================================================
void ArrAllocSglC(size_t *HMat, int ArrSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*ArrSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSgl
///==========================================================
void ArrInitSgl(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	dMat = (float*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglC
///==========================================================
void ArrInitSglC(size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	dMat = (float*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSgl
///==========================================================
void ArrFreeSgl(size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	dMat = (float*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglC
///==========================================================
void ArrFreeSglC(size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	dMat = (float*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSgl
///==========================================================
void ArrLoadSgl(float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	dMat = (float*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglC
///==========================================================
void ArrLoadSglC(float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	dMat = (float*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSgl
///==========================================================
void ArrReturnSgl(float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz;
	dMat = (float*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSglC
///==========================================================
void ArrReturnSglC(float *Mat, size_t *HMat, int ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz*2;
	dMat = (float*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat3DAllocSgl
///==========================================================
void Mat3DAllocSgl(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DAllocSglC
///==========================================================
void Mat3DAllocSglC(size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t free,total;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSgl
///==========================================================
void Mat3DInitSgl(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	dMat = (float*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSglC
///==========================================================
void Mat3DInitSglC(size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	dMat = (float*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSgl
///==========================================================
void Mat3DFreeSgl(size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	dMat = (float*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSglC
///==========================================================
void Mat3DFreeSglC(size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	dMat = (float*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSgl
///==========================================================
void Mat3DLoadSgl(float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	dMat = (float*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSglC
///==========================================================
void Mat3DLoadSglC(float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	dMat = (float*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSgl
///==========================================================
void Mat3DReturnSgl(float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	dMat = (float*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSglC
///==========================================================
void Mat3DReturnSglC(float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	dMat = (float*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
