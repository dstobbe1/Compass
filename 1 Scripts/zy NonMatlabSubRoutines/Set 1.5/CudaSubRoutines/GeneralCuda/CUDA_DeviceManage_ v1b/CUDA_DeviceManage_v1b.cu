
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		- Recompile with CUDA 7.5
///     - add 'cudaDeviceSynchronize()'
///==========================================================

extern "C" void CUDAselect(int device, char *Error);
extern "C" void CUDAreset(char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAselect(int device, char *Error){ 
	const char* Error0; 
	hipSetDevice(device);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// CUDA reset
///==========================================================
void CUDAreset(char *Error){ 
	const char* Error0; 
	hipDeviceReset();
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
