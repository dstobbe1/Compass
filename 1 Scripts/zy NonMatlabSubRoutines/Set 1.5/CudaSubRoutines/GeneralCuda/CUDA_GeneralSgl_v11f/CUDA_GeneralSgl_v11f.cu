
#include <hip/hip_runtime.h>
///==========================================================
/// (v11f)
///     - v11 now means for Cuda 11
///		- Drop SampDat
///     - Simplify 'ArrLoadSglOne'
///==========================================================

extern "C" void CudaDeviceWait(size_t *GpuNum, char *Error);
extern "C" void ArrAllocSglAll(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrAllocSglAllC(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAll(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAllC(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrFreeSglAll(size_t *GpuTot, size_t *HMat, char *Error);
extern "C" void ArrFreeSglAllC(size_t *GpuTot, size_t *HMat, char *Error);
extern "C" void ArrLoadSglAll(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglAllAsync(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglAllC(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOne(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOneC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOneAsyncC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOne(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOneC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOneAsyncC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglAll(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglAllC(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);


///==========================================================
/// CudaDeviceWait
///==========================================================
void CudaDeviceWait(size_t *GpuNum, char *Error){
    const char* Error0; 
    hipSetDevice(GpuNum[0]);
	hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocSglAll
///==========================================================
void ArrAllocSglAll(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error){
    const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);                           // implicitly synchronous
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();
}

///==========================================================
/// ArrAllocSglAllC
///==========================================================
void ArrAllocSglAllC(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAll
///==========================================================
void ArrInitSglAll(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAllC
///==========================================================
void ArrInitSglAllC(size_t *GpuTot, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSglAll
///==========================================================
void ArrFreeSglAll(size_t *GpuTot, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglAllC
///==========================================================
void ArrFreeSglAllC(size_t *GpuTot, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAll
///==========================================================
void ArrLoadSglAll(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAllAsync
///==========================================================
void ArrLoadSglAllAsync(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrLoadSglAllC
///==========================================================
void ArrLoadSglAllC(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrLoadSglOne
///==========================================================
void ArrLoadSglOne(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    
    hipSetDevice(GpuNum[0]);
	dMat = (float*)HMat[GpuNum[0]];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglOneC
///==========================================================
void ArrLoadSglOneC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
    
    hipSetDevice(GpuNum[0]);
	dMat = (float*)HMat[GpuNum[0]];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrLoadSglOneAsyncC
///==========================================================
void ArrLoadSglOneAsyncC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
    
    hipSetDevice(GpuNum[0]);
	dMat = (float*)HMat[GpuNum[0]];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);	
}

///==========================================================
/// ArrReturnSglOne
///==========================================================
void ArrReturnSglOne(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	
    hipSetDevice(GpuNum[0]);
    dMat = (float*)HMat[GpuNum[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglOneC
///==========================================================
void ArrReturnSglOneC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
    hipSetDevice(GpuNum[0]);
    dMat = (float*)HMat[GpuNum[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSglOneAsyncC
///==========================================================
void ArrReturnSglOneAsyncC(size_t *GpuNum, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
    hipSetDevice(GpuNum[0]);
    dMat = (float*)HMat[GpuNum[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);	
}

///==========================================================
/// ArrReturnSglAll
///==========================================================
void ArrReturnSglAll(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	}
    	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglAllC
///==========================================================
void ArrReturnSglAllC(size_t *GpuTot, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
	for(int n=0;n<GpuTot[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	}
    	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}



