
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- 
///==========================================================

extern "C" void ArrMSAllocDbl(size_t *HMat, int ArrSz, int nVols, size_t *Tst, char *Error);
extern "C" void ArrMSAllocDblC(size_t *HMat, int ArrSz, int nVols, size_t *Tst, char *Error);
extern "C" void ArrMSInitDbl(size_t *HMat, int ArrSz, int nVols, char *Error);
extern "C" void ArrMSInitDblC(size_t *HMat, int ArrSz, int nVols, char *Error);
extern "C" void ArrMSFreeDbl(size_t *HMat, char *Error);
extern "C" void ArrMSFreeDblC(size_t *HMat, char *Error);
extern "C" void ArrMSLoadDbl(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error);
extern "C" void ArrMSLoadDblC(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error);
extern "C" void ArrMSReturnDbl(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error);
extern "C" void ArrMSReturnDblC(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error);

extern "C" void Mat2DMSAllocDbl(size_t *HMat, int MatSz, int nVols, size_t *Tst, char *Error);
extern "C" void Mat2DMSAllocDblC(size_t *HMat, int MatSz, int nVols, size_t *Tst, char *Error);
extern "C" void Mat2DMSInitDbl(size_t *HMat, int MatSz, int nVols, char *Error);
extern "C" void Mat2DMSInitDblC(size_t *HMat, int MatSz, int nVols, char *Error);
extern "C" void Mat2DMSFreeDbl(size_t *HMat, char *Error);
extern "C" void Mat2DMSFreeDblC(size_t *HMat, char *Error);
extern "C" void Mat2DMSLoadDbl(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error);
extern "C" void Mat2DMSLoadDblC(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error);
extern "C" void Mat2DMSReturnDbl(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error);
extern "C" void Mat2DMSReturnDblC(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error);

///==========================================================
/// ArrAllocDbl
///==========================================================
void ArrMSAllocDbl(size_t *HMat, int ArrSz, int nVols, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz*nVols;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrAllocDblC
///==========================================================
void ArrMSAllocDblC(size_t *HMat, int ArrSz, int nVols, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*ArrSz*nVols*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDbl
///==========================================================
void ArrMSInitDbl(size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitDblC
///==========================================================
void ArrMSInitDblC(size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeDbl
///==========================================================
void ArrMSFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeDblC
///==========================================================
void ArrMSFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDbl
///==========================================================
void ArrMSLoadDbl(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadDblC
///==========================================================
void ArrMSLoadDblC(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDbl
///==========================================================
void ArrMSReturnDbl(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnDblC
///==========================================================
void ArrMSReturnDblC(double *Mat, size_t *HMat, int ArrSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*ArrSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


///==========================================================
/// Mat2DAllocDbl
///==========================================================
void Mat2DMSAllocDbl(size_t *HMat, int MatSz, int nVols, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DAllocDblC
///==========================================================
void Mat2DMSAllocDblC(size_t *HMat, int MatSz, int nVols, size_t *Tst, char *Error){
	const char* Error0; 
	double *dMat;
	size_t free,total;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols*2;
	hipMalloc(&dMat,MatMem);
	*HMat = (size_t)dMat;
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipMemGetInfo(&free,&total);
	Tst[0] = total;
	Tst[1] = free;
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DInitDbl
///==========================================================
void Mat2DMSInitDbl(size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DInitDblC
///==========================================================
void Mat2DMSInitDblC(size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemset(dMat,0,MatMem);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DFreeDbl
///==========================================================
void Mat2DMSFreeDbl(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DFreeDblC
///==========================================================
void Mat2DMSFreeDblC(size_t *HMat, char *Error){
	const char* Error0; 
	double *dMat;
	dMat = (double*)*HMat;
	hipFree(dMat);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DLoadDbl
///==========================================================
void Mat2DMSLoadDbl(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DLoadDblC
///==========================================================
void Mat2DMSLoadDblC(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemcpy(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DReturnDbl
///==========================================================
void Mat2DMSReturnDbl(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat2DReturnDblC
///==========================================================
void Mat2DMSReturnDblC(double *Mat, size_t *HMat, int MatSz, int nVols, char *Error){
	const char* Error0; 
	double *dMat;
	size_t MatMem = sizeof(double)*MatSz*MatSz*nVols*2;
	dMat = (double*)*HMat;
	hipMemcpy(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
