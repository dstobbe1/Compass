
#include <hip/hip_runtime.h>
///==========================================================
/// (v1a)
///		- 
///==========================================================

extern "C" void CUDAselect(int device, char *Error);
extern "C" void CUDAreset(char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAselect(int device, char *Error){ 
	const char* Error0; 
	hipSetDevice(device);
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// CUDA reset
///==========================================================
void CUDAreset(char *Error){ 
	const char* Error0; 
	hipDeviceReset();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}
