///==========================================================
/// CudaDataChunkLoad
///     DatInfo
///         0) Wid          (5)             - Kx,Ky,Kz,Comp,Dat)
///         1) Len          (np*chunk*2)    - the 2 is for complex
///         2) ChanTot      (total channels)
///         3) ChanStart    (channel to start)
///         4) ChanWrite    (channels to write)
///==========================================================
void CudaDataChunkLoad(size_t *Device, float *Mat, size_t *HMat, size_t *MatLen){

	float *dMat;
	size_t MatMem = sizeof(float)*MatLen[0];
    hipSetDevice(Device[0]);    
    dMat = (float*)HMat[n];
    hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
}


