
#include <hip/hip_runtime.h>
///==========================================================
/// (v1e)
///		- Renaming to make sense
///     - Drop Mat
///     - SampDat Copy
///==========================================================

extern "C" void CUDAcount(int *Count, char *Error);
extern "C" void ArrAllocSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrAllocSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrFreeSglAll(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrFreeSglAllC(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrLoadSglAll(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error);
extern "C" void ArrLoadSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error);
extern "C" void ArrReturnSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void SampDatLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *DataInfo, char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAcount(int *Count, char *Error){ 
	const char* Error0; 
	hipGetDeviceCount(Count);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocSglAll
///==========================================================
void ArrAllocSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
    const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();
}

///==========================================================
/// ArrAllocSglAllC
///==========================================================
void ArrAllocSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAll
///==========================================================
void ArrInitSglAll(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglAllC
///==========================================================
void ArrInitSglAllC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSglAll
///==========================================================
void ArrFreeSglAll(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglAllC
///==========================================================
void ArrFreeSglAllC(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAll
///==========================================================
void ArrLoadSglAll(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglAllC
///==========================================================
void ArrLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrLoadSglOne
///==========================================================
void ArrLoadSglOne(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
    dMat = dMat + ProjInc[0]*ArrSz[0];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglOneC
///==========================================================
void ArrLoadSglOneC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
    dMat = dMat + ProjInc[0]*ArrSz[0]*2;
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSglOne
///==========================================================
void ArrReturnSgl(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglOneC
///==========================================================
void ArrReturnSglC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// SampDatLoadSglAllC
///     DatInfo
///         0) Nro
///         1) Nrcvr
///         2) Nproj    (number of projections in chunk)
///         3) Sro      (start)
///         4) Srcvr
///         5) Sproj
///==========================================================
void SampDatLoadSglAllC(size_t *Count, float *Mat, size_t *HMat, size_t *DataInfo, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*DataInfo[0]*2;
    for(int m=DataInfo[5];m<DataInfo[2];m++){
        for(int n=0;n<Count[0];n++){
            hipSetDevice(n);
            Mat = Mat + ((DataInfo[4]+n)*DataInfo[0] + m*DataInfo[0]*DataInfo[1] + DataInfo[3])*2; 
            dMat = (float*)HMat[n] + m*DataInfo[0]*2;
            hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
        }
    }
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}


