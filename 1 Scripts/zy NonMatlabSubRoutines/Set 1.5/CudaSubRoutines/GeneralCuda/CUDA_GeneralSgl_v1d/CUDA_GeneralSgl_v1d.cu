
#include <hip/hip_runtime.h>
///==========================================================
/// (v1d)
///		- Multi GPUs Supported Within
///     - Update for New Object-Based recon
///==========================================================

extern "C" void CUDAcount(int *Count, char *Error);
//extern "C" void CUDAselect(int device, char *Error);
//extern "C" void CUDAreset(char *Error);

extern "C" void ArrAllocSgl(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrAllocSglC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSgl(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrInitSglC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrFreeSgl(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrFreeSglC(size_t *Count, size_t *HMat, char *Error);
extern "C" void ArrLoadSglSync(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSglSyncC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrLoadSgl(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error);
extern "C" void ArrLoadSglC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error);
extern "C" void ArrReturnSgl(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);
extern "C" void ArrReturnSglSyncC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error);

extern "C" void Mat3DAllocSgl(size_t *Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DAllocSglC(size_t *Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitSgl(size_t *Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DInitSglC(size_t *Count, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DFreeSgl(size_t *Count,size_t *HMat, char *Error);
extern "C" void Mat3DFreeSglC(size_t *Count, size_t *HMat, char *Error);
extern "C" void Mat3DLoadSgl(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DLoadSglC(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSgl(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error);
extern "C" void Mat3DReturnSglC(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error);

///==========================================================
/// CUDA select
///==========================================================
void CUDAcount(int *Count, char *Error){ 
	const char* Error0; 
	hipGetDeviceCount(Count);
    hipDeviceSynchronize();
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
}

///==========================================================
/// ArrAllocSgl
///==========================================================
void ArrAllocSgl(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
    const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();
}

///==========================================================
/// ArrAllocSglC
///==========================================================
void ArrAllocSglC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSgl
///==========================================================
void ArrInitSgl(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrInitSglC
///==========================================================
void ArrInitSglC(size_t *Count, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrFreeSgl
///==========================================================
void ArrFreeSgl(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrFreeSglC
///==========================================================
void ArrFreeSglC(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglSync
///==========================================================
void ArrLoadSglSync(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglSyncC
///==========================================================
void ArrLoadSglSyncC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrLoadSgl
///==========================================================
void ArrLoadSgl(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
    dMat = dMat + ProjInc[0]*ArrSz[0];
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();	
}

///==========================================================
/// ArrLoadSglC
///==========================================================
void ArrLoadSglC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, size_t *ProjInc, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
    
    hipSetDevice(Count[0]);
	dMat = (float*)HMat[Count[0]];
    dMat = dMat + ProjInc[0]*ArrSz[0]*2;
	hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSgl
///==========================================================
void ArrReturnSgl(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0];
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();				
	}

///==========================================================
/// ArrReturnSglC
///==========================================================
void ArrReturnSglC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// ArrReturnSglSyncC
///==========================================================
void ArrReturnSglSyncC(size_t *Count, float *Mat, size_t *HMat, size_t *ArrSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*ArrSz[0]*2;
	
	for(int n=0;n<Count[0];n++){
        hipSetDevice(n);
        dMat = (float*)HMat[n];
        hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);
    }
	
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}






///==========================================================
/// Mat3DAllocSgl
///==========================================================
void Mat3DAllocSgl(size_t *Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DAllocSglC
///==========================================================
void Mat3DAllocSglC(size_t *Count, size_t *HMat, int MatSz, size_t *Tst, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		hipMalloc(&dMat,MatMem);
		HMat[n] = (size_t)dMat;
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSgl
///==========================================================
void Mat3DInitSgl(size_t *Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DInitSglC
///==========================================================
void Mat3DInitSglC(size_t *Count, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemsetAsync(dMat,0,MatMem);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSgl
///==========================================================
void Mat3DFreeSgl(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DFreeSglC
///==========================================================
void Mat3DFreeSglC(size_t *Count, size_t *HMat, char *Error){
	const char* Error0; 
	float *dMat;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipFree(dMat);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSgl
///==========================================================
void Mat3DLoadSgl(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DLoadSglC
///==========================================================
void Mat3DLoadSglC(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;
	for(int n=0;n<Count[0];n++){
		hipSetDevice(n);
		dMat = (float*)HMat[n];
		hipMemcpyAsync(dMat,Mat,MatMem,hipMemcpyHostToDevice);
	}
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSgl
///==========================================================
void Mat3DReturnSgl(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz;

    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}

///==========================================================
/// Mat3DReturnSglC
///==========================================================
void Mat3DReturnSglC(size_t *Count, float *Mat, size_t *HMat, int MatSz, char *Error){
	const char* Error0; 
	float *dMat;
	size_t MatMem = sizeof(float)*MatSz*MatSz*MatSz*2;

    hipSetDevice(Count[0]);
    dMat = (float*)HMat[Count[0]];
    hipMemcpyAsync(Mat,dMat,MatMem,hipMemcpyDeviceToHost);

	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	hipDeviceSynchronize();		
}
