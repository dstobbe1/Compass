///==========================================================
/// (v1a)
///		
///==========================================================

#include "hipfft/hipfft.h"

extern "C" void FFT3D(float* Im, float* kDat, int MatSz, int* Tst, char* Error);

///=====================================================
/// Code Entry
///=====================================================
void FFT3D(float* Im, float* kDat, int MatSz, int* Tst, char* Error)
{

//----------------------------------------------
// General Setup
//----------------------------------------------
const char* Error0; 

//----------------------------------------------
// FFT Setup
//----------------------------------------------
hipfftHandle plan;
hipfftComplex *dIm, *dkDat;

//----------------------------------------------
// Allocate Memory
//----------------------------------------------
size_t MatMem = sizeof(hipfftComplex)*MatSz*MatSz*MatSz;
hipMalloc((void**)&dIm, MatMem);
hipMalloc((void**)&dkDat, MatMem);
if (hipGetLastError() != hipSuccess){
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	return;
}

//----------------------------------------------
// Copy/Set Memory
//----------------------------------------------
hipMemcpy(dIm,Im,MatMem,hipMemcpyHostToDevice);
hipMemset(dkDat,0,MatMem);

//----------------------------------------------
// Create a 3D FFT plan
//----------------------------------------------
hipfftPlan3d(&plan,MatSz,MatSz,MatSz,HIPFFT_C2C);
if (hipGetLastError() != hipSuccess){
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	return;
}

//----------------------------------------------
// Transform
//----------------------------------------------
hipfftExecC2C(plan,dIm,dkDat,HIPFFT_FORWARD);
if (hipGetLastError() != hipSuccess){
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	return;
}

//----------------------------------------------
// Wait for device to finish
//----------------------------------------------
hipDeviceSynchronize();
if (hipGetLastError() != hipSuccess){
	Error0 = hipGetErrorString(hipGetLastError());
	strcpy(Error,Error0);
	return;
}

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(kDat,dkDat,MatMem,hipMemcpyDeviceToHost);

//----------------------------------------------
// Release
//----------------------------------------------
hipfftDestroy(plan);
hipFree(dIm);
hipFree(dkDat);

}

