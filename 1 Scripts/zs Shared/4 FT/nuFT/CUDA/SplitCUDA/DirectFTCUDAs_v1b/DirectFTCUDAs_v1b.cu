
#include <hip/hip_runtime.h>
///==========================================================
/// (v1b)
///		- update for Titan Black
///==========================================================

extern "C" void DirectFT(float* Im, float* kLoc, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int SegX, int SegY, int SegZ, int* Tst, char* Error);

///=====================================================
/// Kernel
///=====================================================
__global__ void FT(float* Im, float* kLoc, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int SegX, int SegY, int SegZ)
{
int k,x,y,z;
float kR,kI;
float pi;
pi = 3.141592;
kR = 0;
kI = 0;
k = blockDim.x*blockIdx.x + threadIdx.x;
if (k < kLen)
    {
    for(x=0; x<8; x++) {
        for(y=0; y<8; y++) {
            for(z=0; z<8; z++) { 
                kR = kR + Im[(x)+(y*8)+(z*64)]*cosf(2*pi*((kLoc[k*3+0]*(SegX+x)/X)+(kLoc[k*3+1]*(SegY+y)/Y)+(kLoc[k*3+2]*(SegZ+z)/Z)));
                kI = kI - Im[(x)+(y*8)+(z*64)]*sinf(2*pi*((kLoc[k*3+0]*(SegX+x)/X)+(kLoc[k*3+1]*(SegY+y)/Y)+(kLoc[k*3+2]*(SegZ+z)/Z)));
                }
            }
        }  
    }
kValR[k] = kR;
kValI[k] = kI; 
}

///=====================================================
/// Code Entry
///=====================================================
void DirectFT(float* Im, float* kLoc, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int SegX, int SegY, int SegZ, int* Tst, char* Error) 
{

//----------------------------------------------
// Test for Device
//----------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);

//----------------------------------------------
// Allocate Device Memory 
//----------------------------------------------
size_t kSize = kLen*sizeof(float);
size_t ImSize = 512*sizeof(float);
float *dIm,*dkLoc,*dkValR,*dkValI;

hipMalloc((void**)&dIm,ImSize);
hipMalloc((void**)&dkLoc,3*kSize);
hipMalloc((void**)&dkValR,kSize);
hipMalloc((void**)&dkValI,kSize);

//----------------------------------------------
// Test Memory Availability
//----------------------------------------------
size_t free,total;
hipMemGetInfo(&free,&total);
Tst[0] = sizeof(float);
Tst[1] = sizeof(int);
Tst[2] = total;
Tst[3] = free;

//----------------------------------------------
// Copy/Set Memory
//----------------------------------------------
hipMemcpy(dIm,Im,ImSize,hipMemcpyHostToDevice);
hipMemcpy(dkLoc,kLoc,3*kSize,hipMemcpyHostToDevice);
hipMemset(dkValR,0,kSize);
hipMemset(dkValI,0,kSize);

//----------------------------------------------
// Invoke kernel
//----------------------------------------------
int tpb = 960;                                                          // possible to go up to 1024 (compute capability 3.5)
int bpg = int(ceil(float(kLen)/float(tpb)));                            // 15 multiprocessors with 192 cores each on Titan Black (use 192x3)
FT<<<bpg,tpb>>>(dIm,dkLoc,dkValR,dkValI,X,Y,Z,kLen,SegX,SegY,SegZ);

const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);
//cudaThreadSynchronize();

//----------------------------------------------
// Copy Back to Host
//----------------------------------------------
hipMemcpy(kValR,dkValR,kSize,hipMemcpyDeviceToHost);
hipMemcpy(kValI,dkValI,kSize,hipMemcpyDeviceToHost);

//----------------------------------------------
// Free device memory
//----------------------------------------------
hipFree(dkLoc);
hipFree(dIm);
hipFree(dkValR);
hipFree(dkValI);

}

