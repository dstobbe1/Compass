
#include <hip/hip_runtime.h>
///==========================================================
/// (v2b)
///		- Split into 2 kernels 
///			(works for image matrix x64)
///			- to do bigger need to increase GRDDIM 
///			- however, would need to split FTfin
///==========================================================

extern "C" void DirectFT(float* Im, float* kLoc, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int* Tst, char* Error);


#define BLKDIM 8
#define GRDDIM 8
#define GRDTOT GRDDIM*GRDDIM*GRDDIM

///=====================================================
/// FT (kernel)
///=====================================================
__global__ void FT(float* Im, float* kLocX, float* kLocY, float* kLocZ, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int tilenum)
{	

float pi = 3.141592;
__shared__ float tIm[BLKDIM][BLKDIM][BLKDIM];

int tx = threadIdx.x;
int ty = threadIdx.y;
int tz = threadIdx.z;
int bx = blockIdx.x;
int by = blockIdx.y;
int bz = blockIdx.z;
int tbx = blockIdx.x * BLKDIM;
int tby = blockIdx.y * BLKDIM;
int tbz = blockIdx.z * BLKDIM;
int ix = tbx + tx;
int iy = tby + ty;
int iz = tbz + tz;
int bnum = bx + by*gridDim.x + bz*gridDim.x*gridDim.y;
tIm[tx][ty][tz] = Im[(ix)+(iy*X)+(iz*X*Y)];	

int n = (tx)+(ty*BLKDIM)+(tz*BLKDIM*BLKDIM);	
int k = BLKDIM*BLKDIM*BLKDIM*tilenum + n;				
float tkLocX = kLocX[k];
float tkLocY = kLocY[k];
float tkLocZ = kLocZ[k];
float tkValR = 0;
float tkValI = 0;
__syncthreads();
	
for(int z=0; z<BLKDIM; z++) {     
	for(int y=0; y<BLKDIM; y++) {
		for(int x=0; x<BLKDIM; x++) {
			tkValR += tIm[x][y][z]*cosf(2*pi*((tkLocX*(tbx+x)/X)+(tkLocY*(tby+y)/Y)+(tkLocZ*(tbz+z)/Z)));			
			tkValI -= tIm[x][y][z]*sinf(2*pi*((tkLocX*(tbx+x)/X)+(tkLocY*(tby+y)/Y)+(tkLocZ*(tbz+z)/Z)));
			__syncthreads();
			}
		}
	}

if (k < kLen) {		
	kValR[k*GRDTOT + bnum] = tkValR;
	kValI[k*GRDTOT + bnum] = tkValI;
	}	
}

///=====================================================
/// FTfin (kernel)
///=====================================================
__global__ void FTfin(float* kValR, float* kValI, float* kValR0, float* kValI0)
{	
__shared__ float tkValR[GRDTOT];
__shared__ float tkValI[GRDTOT];
float accR = 0;
float accI = 0;
int k = blockIdx.x;
int n = threadIdx.x;
int a = k*GRDTOT+n;
tkValR[n] = kValR0[a];
tkValI[n] = kValI0[a];

for(int i=0; i<GRDTOT; i++) { 
	accR += tkValR[i];
	accI += tkValI[i];
	}
	
kValR[k] = accR;
kValI[k] = accI;
}
	
///=====================================================
/// Code Entry
///=====================================================
void DirectFT(float* Im, float* kLoc, float* kValR, float* kValI, int X, int Y, int Z, int kLen, int* Tst, char* Error) 
{

//----------------------------------------------
// Test for Device
//----------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             

//----------------------------------------------
// Allocate Device Memory 
//----------------------------------------------
size_t kSize = kLen*sizeof(float);
size_t ImSize = X*Y*Z*sizeof(float);
float *dIm,*dkLocX,*dkLocY,*dkLocZ,*dkValR0,*dkValI0,*dkValR,*dkValI;

hipMalloc((void**)&dIm,ImSize);
hipMalloc((void**)&dkLocX,kSize);
hipMalloc((void**)&dkLocY,kSize);
hipMalloc((void**)&dkLocZ,kSize);
hipMalloc((void**)&dkValR0,kSize*GRDTOT);
hipMalloc((void**)&dkValI0,kSize*GRDTOT);
hipMalloc((void**)&dkValR,kSize);
hipMalloc((void**)&dkValI,kSize);

//----------------------------------------------
// Test Memory Availability
//----------------------------------------------
size_t free,total;
hipMemGetInfo(&free,&total);
Tst[0] = sizeof(float);
Tst[1] = sizeof(int);
Tst[2] = total;
Tst[3] = free;
Tst[10] = X*Y*Z;
Tst[11] = kLen;

//----------------------------------------------
// Copy/Set Memory
//----------------------------------------------
hipMemcpy(dIm,Im,ImSize,hipMemcpyHostToDevice);
hipMemcpy(dkLocX,kLoc,kSize,hipMemcpyHostToDevice);
hipMemcpy(dkLocY,kLoc+kLen,kSize,hipMemcpyHostToDevice);
hipMemcpy(dkLocZ,kLoc+2*kLen,kSize,hipMemcpyHostToDevice);
hipMemset(dkValR0,0,kSize*GRDTOT);
hipMemset(dkValI0,0,kSize*GRDTOT);
hipMemset(dkValR,0,kSize);
hipMemset(dkValI,0,kSize);

//----------------------------------------------
// Kernel Props
//----------------------------------------------											                        
Tst[4] = BLKDIM;
Tst[5] = GRDDIM;
dim3 tpbmat(BLKDIM,BLKDIM,BLKDIM);  									// possible to go up to 1024. Should be multiple of warp_size=32.
dim3 bpgmat(GRDDIM,GRDDIM,GRDDIM);

//----------------------------------------------
// Invoke kernel
//----------------------------------------------
//int tilenum = 0;
//FT<<<bpgmat,tpbmat>>>(dIm,dkLocX,dkLocY,dkLocZ,dkValR0,dkValI0,X,Y,Z,kLen,tilenum);

int ntiles = int(ceil(float(kLen)/float(BLKDIM*BLKDIM*BLKDIM))); 
Tst[6] = ntiles;
for (int tilenum=0; tilenum<ntiles; tilenum++){
	FT<<<bpgmat,tpbmat>>>(dIm,dkLocX,dkLocY,dkLocZ,dkValR0,dkValI0,X,Y,Z,kLen,tilenum);
	}

//----------------------------------------------
// Invoke kernel2
//----------------------------------------------
int tpb = GRDTOT;
int bpg = kLen;
FTfin<<<bpg,tpb>>>(dkValR,dkValI,dkValR0,dkValI0);
	
//----------------------------------------------
// Pick Up Error
//----------------------------------------------
const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//----------------------------------------------
// Copy Back to Host
//----------------------------------------------
hipMemcpy(kValR,dkValR,kSize,hipMemcpyDeviceToHost);
hipMemcpy(kValI,dkValI,kSize,hipMemcpyDeviceToHost);

//----------------------------------------------
// Free device memory
//----------------------------------------------
hipFree(dIm);
hipFree(dkLocX);
hipFree(dkLocY);
hipFree(dkLocZ);
hipFree(dkValR);
hipFree(dkValI);
hipFree(dkValR0);
hipFree(dkValI0);

}

