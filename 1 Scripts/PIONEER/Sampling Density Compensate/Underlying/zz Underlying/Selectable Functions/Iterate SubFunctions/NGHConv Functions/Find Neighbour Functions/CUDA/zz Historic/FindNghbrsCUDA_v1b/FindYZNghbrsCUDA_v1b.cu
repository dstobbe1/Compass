
#include <hip/hip_runtime.h>
//======================================================
// Find Y Nghbrs
//======================================================

extern "C" void FindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                         int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen, int* Tst, char* Error, int memtst);


//======================================================
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < Dptlen){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[Dpts[i]]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = (temp);    
                p++;
                }
            }
        }
    Kyindlens[i] = (p);
    }
}

void FindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen, int* Tst, char* Error, int memtst){
        
//-----------------------------------------------------
// testing...
//-----------------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             
size_t free;
size_t total;

//-----------------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-----------------------------------------------------
size_t KySz = npro*nproj*sizeof(float);
size_t KxindsSz = Dptlen*maxkxindlen*sizeof(int);
size_t KyindsSz = Dptlen*maxkyindlen*sizeof(int);
size_t KxindlensSz = Dptlen*sizeof(int);
size_t KyindlensSz = Dptlen*sizeof(int);
size_t DptsSz = Dptlen*sizeof(int);

float *dKy;
int *dKxinds,*dKyinds,*dKxindlens,*dKyindlens,*dDpts;
hipMalloc((void**)&dKy,KySz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);
hipMalloc((void**)&dKyindlens,KyindlensSz);
hipMalloc((void**)&dDpts,DptsSz);

const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

hipMemGetInfo(&free,&total);
Tst[0] = total;
Tst[1] = free;
if ((free == 0) || (memtst == 1)) {
    hipFree(dKy);
    hipFree(dKxinds);
    hipFree(dKyinds);
    hipFree(dKyindlens);
    hipFree(dKxindlens);
    hipFree(dDpts);
    return;
}

hipMemcpy(dKy,Ky,KySz,hipMemcpyHostToDevice);
hipMemcpy(dKxinds,Kxinds,KxindsSz,hipMemcpyHostToDevice);
hipMemcpy(dKxindlens,Kxindlens,KxindlensSz,hipMemcpyHostToDevice);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,KyindlensSz);

Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Invoke kernel
//-----------------------------------------------------
int tpb = 512;                                                                          // possible to go up to 512 - compute capability 1.1 - note only (112 cores) 14 multiprocessors with 8 cores each on 9800 GT 
int bpg = int(ceil(float(npro)*float(nproj)/float(tpb)));                                

Tst[2] = tpb;
Tst[3] = bpg;
Tst[4] = npro;
Tst[5] = nproj;
Tst[6] = maxkxindlen;
Tst[7] = maxkyindlen;
Tst[8] = Dptlen;

KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,npro,nproj,maxkxindlen,maxkyindlen,W,dDpts,Dptlen);
Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kyinds,dKyinds,KyindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kyindlens,dKyindlens,KyindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKy);
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKyindlens);
hipFree(dKxindlens);
hipFree(dDpts);

}

