
#include <hip/hip_runtime.h>
//======================================================
// Find X Nghbrs
//======================================================

struct cudastruct {
    const char *Name;
    int Gblmem;
};

extern "C" void FindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                         int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen,
                         int* Tst, char* ErrString, cudastruct* CUDA, int memtst);


//======================================================
// ** Kernel Routine **
//      Kx:          Kx values of all data points (nproj x npro)
//      Radinds:     Kx values within radius for readout location (top and bottom) (2 x npro) 
//      Kxinds:      Kx values within W for each data point (maxkxinlen x Dptlen)
//======================================================
 __global__ void KernFindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                              int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
roloc = int(floorf(float(Dpts[i])/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
if (i < Dptlen){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[Dpts[i]]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = (k+j*nproj);    
                    p++;
                }
            }
        }
    }
    Kxindlens[i] = (p);
}
}

//======================================================
// ** Find X Neighbours **
//
//====================================================== 
void FindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
              int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen,
              int* Tst, char* ErrString, cudastruct* CUDA, int memtst){

//-------------------------------------------
// 'global' variables
//-------------------------------------------   
hipError_t Err; 
const char* ErrString0;
size_t free,total;

//-------------------------------------------
// Device Info
//-------------------------------------------
int device,deviceNum;
struct hipDeviceProp_t deviceProp; 

cudastruct CUDA0;
hipGetDeviceCount(&deviceNum);
Tst[0] = deviceNum;
if (deviceNum < 1) {
    ErrString0 = "No Cuda Device";
    strcpy(ErrString,ErrString0);
    return;
}
hipGetDevice(&device);
Tst[1] = device;
hipGetDeviceProperties(&deviceProp,device);
CUDA0.Name = deviceProp.name;
CUDA0.Gblmem = deviceProp.totalGlobalMem;    
*CUDA = CUDA0;    


//-------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-------------------------------------------
size_t KxSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(int);
size_t KxindsSz = Dptlen*maxkxindlen*sizeof(int);
size_t KxindlensSz = Dptlen*sizeof(int);
size_t DptsSz = Dptlen*sizeof(int);

float *dKx;
int *dRadinds,*dKxinds,*dKxindlens,*dDpts;
hipMalloc((void**)&dKx,KxSz);
hipMalloc((void**)&dRadinds,RadindsSz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);
hipMalloc((void**)&dDpts,DptsSz);

ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

hipMemGetInfo(&free,&total);
Tst[0] = total;
Tst[1] = free;
if ((free == 0) || (memtst == 1)) {
    hipFree(dKx);
    hipFree(dRadinds);
    hipFree(dKxinds);
    hipFree(dKxindlens);
    hipFree(dDpts);
    return;
}

hipMemcpy(dKx,Kx,KxSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,KxindlensSz);

ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

//-------------------------------------------
// Set Up Kernel
//-------------------------------------------
int tpb = 256;                                               // experimental paramater (range 64 - 1024, multiple 32)...                              
int bpg = int(ceil(float(Dptlen)/float(tpb)));

Tst[2] = tpb;
Tst[3] = bpg;
Tst[4] = npro;
Tst[5] = nproj;
Tst[6] = maxkxindlen;
Tst[7] = Dptlen;

if (bpg > 1024){
    ErrString0 = "Block Size Too Large";
    strcpy(ErrString,ErrString0);
    return;
}

//-------------------------------------------
// Invoke kernel
//-------------------------------------------
KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,npro,nproj,maxkxindlen,W,dDpts,Dptlen);
ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kxinds,dKxinds,KxindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kxindlens,dKxindlens,KxindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKx);
hipFree(dRadinds);
hipFree(dKxinds);
hipFree(dKxindlens);
hipFree(dDpts);

}


