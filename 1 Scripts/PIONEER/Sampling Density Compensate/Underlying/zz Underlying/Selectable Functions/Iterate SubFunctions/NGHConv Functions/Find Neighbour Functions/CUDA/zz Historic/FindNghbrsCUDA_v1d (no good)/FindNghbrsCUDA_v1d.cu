#include "hip/hip_runtime.h"
//======================================================
// Find Nghbrs
//======================================================

//#include "string.h"
#include <algorithm>
using namespace std;


extern "C" void FindNgh(float* Kx, float* Ky, float* Kz, int* Radinds, int* NGH, int* NGHlens, int* indlens,
                        int npro, int nproj, float W, int MaxNghbrs, int* Dpts, int Dptlen0,
                        cudastruct* CUDA, char* ErrString, int* TstIn, int* TstOut);

//======================================================
// ** X Kernel Routine **
//      Kx:          Kx values of all data points (nproj x npro)
//      Radinds:     Kx values within radius for readout location (top and bottom) (2 x npro) 
//      Kxinds:      Kx values within W for each data point (maxkxinlen x Dptlen)
//======================================================
 __global__ void KernFindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                              int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen){


float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             


p = 0;
roloc = int(floorf(float(Dpts[i])/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
/*
if (i < Dptlen){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[Dpts[i]]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = (k+j*nproj);    
                    p++;
                }
            }
        }
    }
    Kxindlens[i] = (p);
}
*/
Kxindlens[i] = Radinds[1];
}

//======================================================
// ** Y Kernel Routine **
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                              int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < Dptlen){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[Dpts[i]]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = (temp);    
                p++;
            }
        }
    }
    Kyindlens[i] = (p);
}
} 
 
//======================================================
// ** Find Neighbour Setup **
//====================================================== 
void FindNgh(float* Kx, float* Ky, float* Kz, int* Radinds, int* NGH, int* NGHlens, int* indlens,
            int npro, int nproj, float W, int MaxNghbrs, int* Dpts, int Dptlen0,
            cudastruct* CUDA, char* ErrString, int* TstIn, int* TstOut){

//-------------------------------------------
// 'global' variables
//-------------------------------------------   
const char* ErrString0;
size_t free,total;

//-------------------------------------------
// Copy Data (unfortunately need to do for each call...)
//-------------------------------------------  
size_t KSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(int);
hipMalloc((void**)&dKx,KSz);
hipMalloc((void**)&dKy,KSz);
hipMalloc((void**)&dKz,KSz);
hipMalloc((void**)&dRadinds,RadindsSz);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    strcat(ErrString," - hipMalloc");
    return;
}

hipMemcpy(dKx,Kx,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKy,Ky,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKz,Kz,KSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    strcat(ErrString," - hipMemcpy");
    return;
}

size_t free,total;
hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;





//-------------------------------------------
// Search 'X'
//-------------------------------------------
int *dKxinds,*dKxindlens,*dDpts;
size_t KxindsSz = Dptlen0*MaxNghbrs*sizeof(int);
size_t DptsSz = Dptlen0*sizeof(int);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,DptsSz);
hipMalloc((void**)&dDpts,DptsSz);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,DptsSz);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
int maxkxindlen = MaxNghbrs;

hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

int tpb = 256;                                               // experimental paramater (range 64 - 1024, multiple 32)...                              
int bpg = int(ceil(float(Dptlen0)/float(tpb)));

if (bpg > 1024){
    ErrString0 = "Block Size Too Large";
    strcpy(ErrString,ErrString0);
    return;
}

KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,nproj,maxkxindlen,W,dDpts,Dptlen0);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}
hipMemcpy(indlens,dKxindlens,DptsSz,hipMemcpyDeviceToHost);
//hipMemcpy(Dpts,dDpts,DptsSz,hipMemcpyDeviceToHost);
TstOut[0] = indlens[50];
strcpy(ErrString,ErrString0);
return;












//maxkxindlen = *max_element(dKxindlens,dKxindlens+Dptlen0);
//maxkxindlen = (int)ceil(float(maxkxindlen)*1.05);

//-------------------------------------------
// Search 'Y'
//-------------------------------------------
int *dKyinds,*dKyindlens;
size_t KyindsSz = Dptlen0*maxkxindlen*sizeof(int);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKyindlens,DptsSz);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,DptsSz);
int maxkyindlen = maxkxindlen;

KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,maxkxindlen,maxkyindlen,W,dDpts,Dptlen0);
maxkyindlen = *max_element(dKyindlens,dKyindlens+Dptlen0);
maxkyindlen = (int)ceil(float(maxkyindlen)*1.05);
ErrString0 = hipGetErrorString(hipGetLastError());

//-------------------------------------------
// Search 'Z'
//-------------------------------------------
int *dKzinds,*dKzindlens;
size_t KzindsSz = Dptlen0*maxkyindlen*sizeof(int);
hipMalloc((void**)&dKzinds,KzindsSz);
hipMalloc((void**)&dKzindlens,DptsSz);
hipMemset(dKyinds,0,KzindsSz);
hipMemset(dKyindlens,0,DptsSz);
int maxkzindlen = maxkyindlen;

KernFindYngh<<<bpg,tpb>>>(dKz,dKyinds,dKzinds,dKyindlens,dKzindlens,maxkyindlen,maxkzindlen,W,dDpts,Dptlen0);
maxkzindlen = *max_element(dKzindlens,dKzindlens+Dptlen0);
maxkzindlen = (int)ceil(float(maxkzindlen)*1.05);
ErrString0 = hipGetErrorString(hipGetLastError());

//----------------------------------------------
// Output
//----------------------------------------------
//NGH = dKzinds;                         // possible keep in memory - for doing convolution.
//NGHlens = dKzindlens;
hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;
TstOut[2] = maxkxindlen;
TstOut[3] = maxkyindlen;
TstOut[4] = maxkzindlen;
if (TstIn[0]==0){
    hipMemcpy(NGH,dKzinds,KzindsSz,hipMemcpyDeviceToHost);
}

//----------------------------------------------
// Free Memory
//----------------------------------------------
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKxindlens);
hipFree(dKyindlens);
hipFree(dDpts);

hipFree(dKzinds);                         // possibly keep in memory - for doing convolution.  
hipFree(dKzindlens);
}


