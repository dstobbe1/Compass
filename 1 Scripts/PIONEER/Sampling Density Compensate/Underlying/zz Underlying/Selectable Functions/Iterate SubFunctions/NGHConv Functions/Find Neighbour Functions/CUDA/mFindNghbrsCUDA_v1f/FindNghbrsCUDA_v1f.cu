//======================================================
// Find Nghbrs
//======================================================


#include <hip/hip_runtime.h>
#include <algorithm>
using namespace std;


extern "C" void FindNgh(float* Kx, float* Ky, float* Kz, int* Radinds, int* NGH, int* NGHlens,
                        int npro, int nproj, float W, int MaxXNghbrs, int MaxYNghbrs, int MaxZNghbrs, int* Dpts, int Dptlen0,
                        char* ErrString, int* TstIn, int* TstOut, float xslmult, float yslmult, float zslmult);

//======================================================
// ** X Kernel Routine **
//      Kx:          Kx values of all data points (nproj x npro)
//      Radinds:     Kx values within radius for readout location (top and bottom) (2 x npro) 
//      Kxinds:      Kx values within W for each data point (maxkxinlen x Dptlen)
//======================================================
 __global__ void KernFindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                              int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen){


float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
roloc = int(floorf(float(Dpts[i])/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
if (i < Dptlen){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[Dpts[i]]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = (k+j*nproj);    
                    p++;
                }
            }
        }
    }
    Kxindlens[i] = (p);
}
}

//======================================================
// ** Y Kernel Routine **
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                              int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < Dptlen){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[Dpts[i]]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = (temp);    
                p++;
            }
        }
    }
    Kyindlens[i] = (p);
}
} 
 
//======================================================
// ** Find Neighbour Setup **
//====================================================== 
void FindNgh(float* Kx, float* Ky, float* Kz, int* Radinds, int* NGH, int* NGHlens,
             int npro, int nproj, float W, int MaxXNghbrs, int MaxYNghbrs, int MaxZNghbrs, int* Dpts, int Dptlen0,
             char* ErrString, int* TstIn, int* TstOut, float xslmult, float yslmult, float zslmult){

//-------------------------------------------
// 'global' variables
//-------------------------------------------   
const char* ErrString0;
size_t free,total;

//-------------------------------------------
// Copy Data (unfortunately need to do for each call...)
//-------------------------------------------  
float *dKx,*dKy,*dKz;
int *dRadinds;
size_t KSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(int);
hipMalloc((void**)&dKx,KSz);
hipMalloc((void**)&dKy,KSz);
hipMalloc((void**)&dKz,KSz);
hipMalloc((void**)&dRadinds,RadindsSz);
hipMemcpy(dKx,Kx,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKy,Ky,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKz,Kz,KSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);

//-------------------------------------------
// Data Points to Search Through
//-------------------------------------------
int *dDpts;
size_t DptsSz = Dptlen0*sizeof(int);
hipMalloc((void**)&dDpts,DptsSz);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);

int tpb = 256;                                               // experimental paramater (range 64 - 1024, multiple 32)...                              
int bpg = int(ceil(float(Dptlen0)/float(tpb)));
if (bpg > 1024){
    ErrString0 = "Block Size Too Large";
    strcpy(ErrString,ErrString0);
    return;
}

//-------------------------------------------
// Search 'X'
//-------------------------------------------
int *dKxinds,*dKxindlens;
size_t KxindsSz = Dptlen0*MaxXNghbrs*sizeof(int);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,DptsSz);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,DptsSz);

hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,nproj,MaxXNghbrs,W,dDpts,Dptlen0);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

int maxkxindlen = 0;
hipMemcpy(NGHlens,dKxindlens,DptsSz,hipMemcpyDeviceToHost);
maxkxindlen  = *max_element(NGHlens,NGHlens+Dptlen0);
if (TstIn[0]==1){
    maxkxindlen  = (int)ceil(float(maxkxindlen)*xslmult);
}
TstOut[2] = maxkxindlen;

//-------------------------------------------
// Search 'Y'
//-------------------------------------------
if (TstIn[0]==1){
    MaxYNghbrs = maxkxindlen;
}
int *dKyinds,*dKyindlens;
size_t KyindsSz = Dptlen0*MaxYNghbrs*sizeof(int);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKyindlens,DptsSz);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,DptsSz);

hipMemGetInfo(&free,&total);
TstOut[3] = total;
TstOut[4] = free;
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,MaxXNghbrs,MaxYNghbrs,W,dDpts,Dptlen0);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

int maxkyindlen = 0;
hipMemcpy(NGHlens,dKyindlens,DptsSz,hipMemcpyDeviceToHost);
maxkyindlen = *max_element(NGHlens,NGHlens+Dptlen0);
if (TstIn[0]==1){
    maxkyindlen = (int)ceil(float(maxkyindlen)*yslmult);
}
TstOut[5] = maxkyindlen;

hipFree(dKxinds);
hipFree(dKxindlens);

//-------------------------------------------
// Search 'Z'
//-------------------------------------------
if (TstIn[0]==1){
    MaxZNghbrs = maxkyindlen;
}
int *dKzinds,*dKzindlens;
size_t KzindsSz = Dptlen0*MaxZNghbrs*sizeof(int);
hipMalloc((void**)&dKzinds,KzindsSz);
hipMalloc((void**)&dKzindlens,DptsSz);
hipMemset(dKzinds,0,KzindsSz);
hipMemset(dKzindlens,0,DptsSz);

hipMemGetInfo(&free,&total);
TstOut[6] = total;
TstOut[7] = free;
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

KernFindYngh<<<bpg,tpb>>>(dKz,dKyinds,dKzinds,dKyindlens,dKzindlens,MaxYNghbrs,MaxZNghbrs,W,dDpts,Dptlen0);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    return;
}

int maxkzindlen = 0;
hipMemcpy(NGHlens,dKzindlens,DptsSz,hipMemcpyDeviceToHost);
maxkzindlen = *max_element(NGHlens,NGHlens+Dptlen0);
if (TstIn[0]==1){
    maxkzindlen = (int)ceil(float(maxkzindlen)*zslmult);
}
TstOut[8] = maxkzindlen;

//----------------------------------------------
// Output
//----------------------------------------------
if (TstIn[0]==0){
    hipMemcpy(NGH,dKzinds,KzindsSz,hipMemcpyDeviceToHost);
}

//----------------------------------------------
// Free Memory
//----------------------------------------------
hipFree(dKyinds);
hipFree(dKyindlens);
hipFree(dDpts);
hipFree(dKzinds);                         
hipFree(dKzindlens);
hipFree(dKx);
hipFree(dKy);
hipFree(dKz);
hipFree(dRadinds);
}


