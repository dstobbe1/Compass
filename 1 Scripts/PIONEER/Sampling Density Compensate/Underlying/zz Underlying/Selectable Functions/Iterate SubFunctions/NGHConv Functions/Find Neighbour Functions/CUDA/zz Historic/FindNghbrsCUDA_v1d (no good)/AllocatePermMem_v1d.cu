
#include <hip/hip_runtime.h>
//======================================================
// Allocate 'Permanent' Memory
//======================================================

extern "C" void AllocatePermMem(float* Kx, float* Ky, float* Kz, int* Radinds,
                                int npro, int nproj, char* ErrString, int* TstOut);

extern __device__ float* dKx;
extern __device__ float* dKy;
extern __device__ float* dKz;
extern __device__ int* dRadinds;

//======================================================
// Allocate/Copy/Set Device Memory ('Permanent')
//====================================================== 
void AllocatePermMem(float* Kx, float* Ky, float* Kz, int* Radinds,
                     int npro, int nproj, char* ErrString, int* TstOut){

const char* ErrString0;    
size_t KSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(int);
hipMalloc((void**)&dKx,KSz);
hipMalloc((void**)&dKy,KSz);
hipMalloc((void**)&dKz,KSz);
hipMalloc((void**)&dRadinds,RadindsSz);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    strcat(ErrString," - hipMalloc");
    return;
}

hipMemcpy(dKx,Kx,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKy,Ky,KSz,hipMemcpyHostToDevice);
hipMemcpy(dKz,Kz,KSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);
ErrString0 = hipGetErrorString(hipGetLastError());
if (strcmp(ErrString0,"no error")!=0){
    strcpy(ErrString,ErrString0);
    strcat(ErrString," - hipMemcpy");
    return;
}

size_t free,total;
hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;


}


