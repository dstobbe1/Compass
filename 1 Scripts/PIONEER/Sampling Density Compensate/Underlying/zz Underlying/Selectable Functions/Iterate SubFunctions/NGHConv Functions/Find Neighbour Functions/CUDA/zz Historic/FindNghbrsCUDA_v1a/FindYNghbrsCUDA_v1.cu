
#include <hip/hip_runtime.h>
//======================================================
// Find Y Nghbrs
//======================================================

extern "C" void FindYngh(float* Ky, float* Kxinds, float* Kyinds, float* Kxindlens, float* Kyindlens,
                         int npro, int nproj, int maxkxindlen, int maykyindlen, float W, int dpts, int* Tst, char* Error);


//======================================================
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, float* Kxinds, float* Kyinds, float* Kxindlens, float* Kyindlens,
                              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int dpts){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < dpts){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[i]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = float(temp);    
                 p++;
                }
            }
        }
    Kyindlens[i] = float(p);
    }
}

void FindYngh(float* Ky, float* Kxinds, float* Kyinds, float* Kxindlens, float* Kyindlens,
              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int dpts, int* Tst, char* Error){
        
//-----------------------------------------------------
// testing...
//-----------------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             
size_t free;
size_t total;

//-----------------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-----------------------------------------------------
size_t KySz = npro*nproj*sizeof(float);
size_t KxindsSz = dpts*maxkxindlen*sizeof(float);
size_t KyindsSz = dpts*maxkyindlen*sizeof(float);
size_t KxindlensSz = dpts*sizeof(float);
size_t KyindlensSz = dpts*sizeof(float);

float *dKy,*dKxinds,*dKyinds,*dKxindlens,*dKyindlens;
hipMalloc((void**)&dKy,KySz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);
hipMalloc((void**)&dKyindlens,KyindlensSz);

hipMemGetInfo(&free,&total);
Tst[0] = sizeof(float);
Tst[1] = sizeof(int);
Tst[2] = total;
Tst[3] = free;

hipMemcpy(dKy,Ky,KySz,hipMemcpyHostToDevice);
hipMemcpy(dKxinds,Kxinds,KxindsSz,hipMemcpyHostToDevice);
hipMemcpy(dKxindlens,Kxindlens,KxindlensSz,hipMemcpyHostToDevice);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,KyindlensSz);

const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Invoke kernel
//-----------------------------------------------------
int tpb = 512;                                                                          // possible to go up to 512 - compute capability 1.1
int bpg = int(ceil(float(npro)*float(nproj)/float(tpb)));                               // 14 multiprocessors with 8 cores each on 9800 GT 
KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,npro,nproj,maxkxindlen,maxkyindlen,W,dpts);

Tst[4] = tpb;
Tst[5] = bpg;
Tst[6] = npro;
Tst[7] = nproj;
Tst[8] = maxkyindlen;
Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kyinds,dKyinds,KyindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kyindlens,dKyindlens,KyindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKy);
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKyindlens);
hipFree(dKxindlens);

}

