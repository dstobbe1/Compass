
#include <hip/hip_runtime.h>
//======================================================
// Find X Nghbrs
//======================================================

extern "C" void FindXngh(float* Kx, float* Radinds, float* Kxinds, float* Kxindlens,
                         int npro, int nproj, int maxkxindlen, float W, int dpts, int* Tst, char* Error);


//======================================================
// Kx:          matrix of Kx values of all data points (nproj x npro)
// Radinds:     matrix of Kx values within radius for readout location (top and bottom) (2 x npro) 
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
//======================================================
 __global__ void KernFindXngh(float* Kx, float* Radinds, float* Kxinds, float* Kxindlens,
                              int npro, int nproj, int maxkxindlen, float W, int dpts){

float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
roloc = int(floorf(float(i)/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
if (i < dpts){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[i]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = float(k+j*nproj);    
                     p++;
                    }
                }
            }
        }
    Kxindlens[i] = float(p);
    }
}

void FindXngh(float* Kx, float* Radinds, float* Kxinds, float* Kxindlens,
              int npro, int nproj, int maxkxindlen, float W, int dpts, int* Tst, char* Error){
        
//-----------------------------------------------------
// testing...
//-----------------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             
size_t free;
size_t total;

//-----------------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-----------------------------------------------------
size_t KxSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(float);
size_t KxindsSz = dpts*maxkxindlen*sizeof(float);
size_t KxindlensSz = dpts*sizeof(float);

float *dKx,*dRadinds,*dKxinds,*dKxindlens;
hipMalloc((void**)&dKx,KxSz);
hipMalloc((void**)&dRadinds,RadindsSz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);

hipMemGetInfo(&free,&total);
Tst[0] = sizeof(float);
Tst[1] = sizeof(int);
Tst[2] = total;
Tst[3] = free;

hipMemcpy(dKx,Kx,KxSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,KxindlensSz);

const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Invoke kernel
//-----------------------------------------------------
int tpb = 512;                                                                          // possible to go up to 512 - compute capability 1.1
int bpg = int(ceil(float(npro)*float(nproj)/float(tpb)));                               // 14 multiprocessors with 8 cores each on 9800 GT 
KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,npro,nproj,maxkxindlen,W,dpts);

Tst[4] = tpb;
Tst[5] = bpg;
Tst[6] = npro;
Tst[7] = nproj;
Tst[8] = maxkxindlen;
Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kxinds,dKxinds,KxindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kxindlens,dKxindlens,KxindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKx);
hipFree(dRadinds);
hipFree(dKxinds);
hipFree(dKxindlens);

}

