
#include <hip/hip_runtime.h>
//======================================================
// QueryReset
//======================================================

struct cudastruct {
    const char *Name;
    int Gblmem;
};

extern "C" void QueryReset(cudastruct* CUDA, char* ErrString);

//======================================================
// QueryReset
//====================================================== 
void QueryReset(cudastruct* CUDA, char* ErrString){

const char* ErrString0;
int device,deviceNum;
struct hipDeviceProp_t deviceProp; 
cudastruct CUDA0;

hipDeviceReset();

hipGetDeviceCount(&deviceNum);
if (deviceNum < 1) {
    ErrString0 = "No Cuda Device";
    strcpy(ErrString,ErrString0);
    return;
}

hipGetDevice(&device);
hipGetDeviceProperties(&deviceProp,device);
CUDA0.Name = deviceProp.name;
CUDA0.Gblmem = deviceProp.totalGlobalMem;    
*CUDA = CUDA0;    

}