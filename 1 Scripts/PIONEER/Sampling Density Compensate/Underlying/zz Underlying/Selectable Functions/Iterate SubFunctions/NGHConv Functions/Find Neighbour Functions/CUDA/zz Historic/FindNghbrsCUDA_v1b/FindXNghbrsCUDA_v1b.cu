
#include <hip/hip_runtime.h>
//======================================================
// Find X Nghbrs
//======================================================

extern "C" void FindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                         int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen, int* Tst, char* Error, int memtst);


//======================================================
// Kx:          matrix of Kx values of all data points (nproj x npro)
// Radinds:     matrix of Kx values within radius for readout location (top and bottom) (2 x npro) 
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x Dptlen)
//======================================================
 __global__ void KernFindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                              int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
roloc = int(floorf(float(Dpts[i])/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
if (i < Dptlen){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[Dpts[i]]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = (k+j*nproj);    
                    p++;
                    }
                }
            }
        }
    Kxindlens[i] = (p);
    }
}

void FindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
              int npro, int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen, int* Tst, char* Error, int memtst){
        
//-----------------------------------------------------
// testing...
//-----------------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             
size_t free;
size_t total;

//-----------------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-----------------------------------------------------
size_t KxSz = npro*nproj*sizeof(float);
size_t RadindsSz = npro*2*sizeof(int);
size_t KxindsSz = Dptlen*maxkxindlen*sizeof(int);
size_t KxindlensSz = Dptlen*sizeof(int);
size_t DptsSz = Dptlen*sizeof(int);

float *dKx;
int *dRadinds,*dKxinds,*dKxindlens,*dDpts;
hipMalloc((void**)&dKx,KxSz);
hipMalloc((void**)&dRadinds,RadindsSz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);
hipMalloc((void**)&dDpts,DptsSz);

const char* Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

hipMemGetInfo(&free,&total);
Tst[0] = total;
Tst[1] = free;
if ((free == 0) || (memtst == 1)) {
    hipFree(dKx);
    hipFree(dRadinds);
    hipFree(dKxinds);
    hipFree(dKxindlens);
    hipFree(dDpts);
    return;
}

hipMemcpy(dKx,Kx,KxSz,hipMemcpyHostToDevice);
hipMemcpy(dRadinds,Radinds,RadindsSz,hipMemcpyHostToDevice);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,KxindlensSz);

Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Invoke kernel
//-----------------------------------------------------
int tpb = 512;                                                                          // possible to go up to 512 - compute capability 1.1 - note only (112 cores) 14 multiprocessors with 8 cores each on 9800 GT 
int bpg = int(ceil(float(npro)*float(nproj)/float(tpb)));                               

Tst[2] = tpb;
Tst[3] = bpg;
Tst[4] = npro;
Tst[5] = nproj;
Tst[6] = maxkxindlen;
Tst[7] = Dptlen;

KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,npro,nproj,maxkxindlen,W,dDpts,Dptlen);
Error0 = hipGetErrorString(hipGetLastError());
strcpy(Error,Error0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kxinds,dKxinds,KxindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kxindlens,dKxindlens,KxindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKx);
hipFree(dRadinds);
hipFree(dKxinds);
hipFree(dKxindlens);
hipFree(dDpts);

}

