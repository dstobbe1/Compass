//======================================================
// Find Nghbrs
//======================================================

//#include "string.h"

#include <hip/hip_runtime.h>
#include <algorithm>
using namespace std;

struct cudastruct {
    const char *Name;
    int Gblmem;
};

extern "C" void FindNgh(float* dKx, float* dKy, float* dKz, int* dRadinds,
                         int npro, int nproj, float W, int MaxNghbrs,
                         char* ErrString, int* TstIn, int* TstOut);


//======================================================
// ** X Kernel Routine **
//      Kx:          Kx values of all data points (nproj x npro)
//      Radinds:     Kx values within radius for readout location (top and bottom) (2 x npro) 
//      Kxinds:      Kx values within W for each data point (maxkxinlen x Dptlen)
//======================================================
 __global__ void KernFindXngh(float* Kx, int* Radinds, int* Kxinds, int* Kxindlens,
                              int nproj, int maxkxindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,j,k,p;
int roloc,rob,rot;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
roloc = int(floorf(float(Dpts[i])/float(nproj)));
rob = Radinds[roloc*2];
rot = Radinds[roloc*2+1];
if (i < Dptlen){
    for (j=rob;j<=rot;j++){ 
        for(k=0;k<nproj;k++){                                                         
            dist = fabsf(Kx[k+j*nproj] - Kx[Dpts[i]]);        
            if (dist < W){
            	if (p < maxkxindlen){
                    Kxinds[p+i*maxkxindlen] = (k+j*nproj);    
                    p++;
                }
            }
        }
    }
    Kxindlens[i] = (p);
}
}

//======================================================
// ** Y Kernel Routine **
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                              int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < Dptlen){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[Dpts[i]]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = (temp);    
                p++;
            }
        }
    }
    Kyindlens[i] = (p);
}
} 
 
//======================================================
// ** Find Neighbours **
//====================================================== 
void FindNgh(float* dKx, float* dKy, float* dKz, int* dRadinds,
             int npro, int nproj, float W, int MaxNghbrs,
             char* ErrString, int* TstIn, int* TstOut){

//-------------------------------------------
// 'global' variables
//-------------------------------------------   
hipError_t Err; 
const char* ErrString0;
size_t free,total;

//-------------------------------------------
// Neighbour Search Setup - X
//-------------------------------------------
int start = 0;
int skip = 100;
const int Dptlen0 = 100;
int Dpts[Dptlen0];
size_t DptsSz = Dptlen0*sizeof(int);
for (int n=start;n<Dptlen0;n++){
    Dpts[n] = n*skip;
}           

int *dKxinds,*dKxindlens,*dDpts;
size_t KxindsSz = Dptlen0*MaxNghbrs*sizeof(int);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKxindlens,DptsSz);
hipMalloc((void**)&dDpts,DptsSz);
hipMemset(dKxinds,0,KxindsSz);
hipMemset(dKxindlens,0,DptsSz);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
int maxkxindlen = MaxNghbrs;

int tpb = 256;                                               // experimental paramater (range 64 - 1024, multiple 32)...                              
int bpg = int(ceil(float(Dptlen0)/float(tpb)));

if (bpg > 1024){
    ErrString0 = "Block Size Too Large";
    strcpy(ErrString,ErrString0);
    return;
}

KernFindXngh<<<bpg,tpb>>>(dKx,dRadinds,dKxinds,dKxindlens,nproj,maxkxindlen,W,dDpts,Dptlen0);
maxkxindlen = *max_element(dKxindlens,dKxindlens+Dptlen0);
maxkxindlen = (int)ceil(float(maxkxindlen)*1.05);
ErrString0 = hipGetErrorString(hipGetLastError());

//-------------------------------------------
// Neighbour Search Setup - Y
//-------------------------------------------
int *dKyinds,*dKyindlens;
size_t KyindsSz = Dptlen0*maxkxindlen*sizeof(int);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKyindlens,DptsSz);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,DptsSz);
int maxkyindlen = maxkxindlen;

KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,maxkxindlen,maxkyindlen,W,dDpts,Dptlen0);
maxkyindlen = *max_element(dKyindlens,dKyindlens+Dptlen0);
maxkyindlen = (int)ceil(float(maxkyindlen)*1.05);
ErrString0 = hipGetErrorString(hipGetLastError());

//-------------------------------------------
// Neighbour Search Setup - Z
//-------------------------------------------
int *dKzinds,*dKzindlens;
size_t KzindsSz = Dptlen0*maxkyindlen*sizeof(int);
hipMalloc((void**)&dKzinds,KzindsSz);
hipMalloc((void**)&dKzindlens,DptsSz);
hipMemset(dKyinds,0,KzindsSz);
hipMemset(dKyindlens,0,DptsSz);
int maxkzindlen = maxkyindlen;

KernFindYngh<<<bpg,tpb>>>(dKz,dKyinds,dKzinds,dKyindlens,dKzindlens,maxkyindlen,maxkzindlen,W,dDpts,Dptlen0);
maxkzindlen = *max_element(dKzindlens,dKzindlens+Dptlen0);
maxkzindlen = (int)ceil(float(maxkzindlen)*1.05);
ErrString0 = hipGetErrorString(hipGetLastError());

//-----------------------------------------------------
// Test Memory
//-----------------------------------------------------
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKzinds);
hipFree(dKxindlens);
hipFree(dKyindlens);
hipFree(dKzindlens);
hipFree(dDpts);

float Mem = 150e6;
int Dptlen = (int)ceil((Mem)/(float(maxkxindlen)+float(maxkyindlen)));
DptsSz = Dptlen*sizeof(int);
KxindsSz = Dptlen*maxkxindlen*sizeof(int);
KyindsSz = Dptlen*maxkyindlen*sizeof(int);
KzindsSz = Dptlen*maxkzindlen*sizeof(int);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKzinds,KzindsSz);
hipMalloc((void**)&dKxindlens,DptsSz);
hipMalloc((void**)&dKyindlens,DptsSz);
hipMalloc((void**)&dKzindlens,DptsSz);
hipMalloc((void**)&dDpts,DptsSz);

hipMemGetInfo(&free,&total);
TstOut[0] = total;
TstOut[1] = free;

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKx);
hipFree(dKy);
hipFree(dKz);
hipFree(dRadinds);
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKzinds);
hipFree(dKxindlens);
hipFree(dKyindlens);
hipFree(dKzindlens);
hipFree(dDpts);

}


