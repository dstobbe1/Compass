
#include <hip/hip_runtime.h>
//======================================================
// Find Y Nghbrs
//======================================================

extern "C" void FindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                         int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen,
                         int* Tst, char* ErrString, int memtst);


//======================================================
// Ky:          matrix of Ky values of all data points (nproj x npro)
// Kxinds:      matrix of Kx values within W for each data point (maxkxinlen x dpts)
// Kyinds:      matrix of Ky values within W for each data point (maxkyinlen x dpts)
//======================================================
 __global__ void KernFindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
                              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen){

float dist;
int i,k,p,temp;
i = blockDim.x*blockIdx.x + threadIdx.x;             

p = 0;
if (i < Dptlen){
    for(k=0;k<int(Kxindlens[i]);k++){                                                         
        temp = int(Kxinds[k+i*maxkxindlen]);
        dist = fabsf(Ky[temp] - Ky[Dpts[i]]);        
        if (dist < W){
            if (p < maxkyindlen){
                Kyinds[p+i*maxkyindlen] = (temp);    
                p++;
            }
        }
    }
    Kyindlens[i] = (p);
}
}

//======================================================
// ** Find YZ Neighbours **
//
//====================================================== 
 void FindYngh(float* Ky, int* Kxinds, int* Kyinds, int* Kxindlens, int* Kyindlens,
              int npro, int nproj, int maxkxindlen, int maxkyindlen, float W, int* Dpts, int Dptlen,
              int* Tst, char* ErrString, int memtst){
        
//-----------------------------------------------------
// testing...
//-----------------------------------------------------
//int deviceCount;
//cudaGetDeviceCount(&deviceCount);
//int device = 0;
//cudaDeviceProp deviceProp;
//cudaGetDeviceProperties(&deviceProp, device);             
size_t free;
size_t total;

//-----------------------------------------------------
// Allocate/Copy/Set Device Memory (Global)
//-----------------------------------------------------
size_t KySz = npro*nproj*sizeof(float);
size_t KxindsSz = Dptlen*maxkxindlen*sizeof(int);
size_t KyindsSz = Dptlen*maxkyindlen*sizeof(int);
size_t KxindlensSz = Dptlen*sizeof(int);
size_t KyindlensSz = Dptlen*sizeof(int);
size_t DptsSz = Dptlen*sizeof(int);

float *dKy;
int *dKxinds,*dKyinds,*dKxindlens,*dKyindlens,*dDpts;
hipMalloc((void**)&dKy,KySz);
hipMalloc((void**)&dKxinds,KxindsSz);
hipMalloc((void**)&dKyinds,KyindsSz);
hipMalloc((void**)&dKxindlens,KxindlensSz);
hipMalloc((void**)&dKyindlens,KyindlensSz);
hipMalloc((void**)&dDpts,DptsSz);

const char* ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

hipMemGetInfo(&free,&total);
Tst[0] = total;
Tst[1] = free;
if ((free == 0) || (memtst == 1)) {
    hipFree(dKy);
    hipFree(dKxinds);
    hipFree(dKyinds);
    hipFree(dKyindlens);
    hipFree(dKxindlens);
    hipFree(dDpts);
    return;
}

hipMemcpy(dKy,Ky,KySz,hipMemcpyHostToDevice);
hipMemcpy(dKxinds,Kxinds,KxindsSz,hipMemcpyHostToDevice);
hipMemcpy(dKxindlens,Kxindlens,KxindlensSz,hipMemcpyHostToDevice);
hipMemcpy(dDpts,Dpts,DptsSz,hipMemcpyHostToDevice);
hipMemset(dKyinds,0,KyindsSz);
hipMemset(dKyindlens,0,KyindlensSz);

ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

//-------------------------------------------
// Set Up Kernel
//-------------------------------------------
int tpb = 256;                                               // experimental paramater (range 64 - 1024, multiple 32)...                              
int bpg = int(ceil(float(Dptlen)/float(tpb)));                          

Tst[2] = tpb;
Tst[3] = bpg;
Tst[4] = npro;
Tst[5] = nproj;
Tst[6] = maxkxindlen;
Tst[7] = maxkyindlen;
Tst[8] = Dptlen;

if (bpg > 1024){
    ErrString0 = "Block Size Too Large";
    strcpy(ErrString,ErrString0);
    return;
}

//-------------------------------------------
// Invoke kernel
//-------------------------------------------
KernFindYngh<<<bpg,tpb>>>(dKy,dKxinds,dKyinds,dKxindlens,dKyindlens,npro,nproj,maxkxindlen,maxkyindlen,W,dDpts,Dptlen);
ErrString0 = hipGetErrorString(hipGetLastError());
strcpy(ErrString,ErrString0);

//-----------------------------------------------------
// Copy Back to Host
//-----------------------------------------------------
hipMemcpy(Kyinds,dKyinds,KyindsSz,hipMemcpyDeviceToHost);
hipMemcpy(Kyindlens,dKyindlens,KyindlensSz,hipMemcpyDeviceToHost);

//-----------------------------------------------------
// Free device memory
//-----------------------------------------------------
hipFree(dKy);
hipFree(dKxinds);
hipFree(dKyinds);
hipFree(dKyindlens);
hipFree(dKxindlens);
hipFree(dDpts);

}

